#include "hip/hip_runtime.h"
#include <tune_quda.h>
#include <clover_field_order.h>

namespace quda {

  using namespace clover;

#ifdef GPU_CLOVER_DIRAC

  template <typename real, typename Clover>
  struct CloverRhoArg  {
    Clover clover;
    real rho;
    CloverRhoArg(Clover &clover, real rho) : clover(clover), rho(rho) {}
  };

  template <int nSpin, int nColor, typename Arg>
  __global__ void cloverRhoKernel(Arg arg) {  

    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    if (x_cb >= arg.clover.volumeCB) return;
    int parity = blockIdx.y*blockDim.y + threadIdx.y;
    int chirality = blockIdx.z*blockDim.z + threadIdx.z;

    constexpr int N = nColor*nSpin/2;
    HMatrix<decltype(arg.rho),N> A = arg.clover(x_cb, parity, chirality);
    for (int i=0; i<N; i++) A(i,i) += arg.rho;
    arg.clover(x_cb, parity, chirality) = A;

  }

  template <int nSpin, int nColor, typename Arg>
  class CloverRho : TunableVectorYZ {
    Arg arg;
    const CloverField &meta; // used for meta data only

  private:
    bool tuneSharedBytes() const { return false; } // Don't tune the shared memory
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return arg.clover.volumeCB; }

  public:
    CloverRho(Arg &arg, const CloverField &meta) : TunableVectorYZ(2,2), arg(arg), meta(meta) {
      writeAuxString("_");
    }
    virtual ~CloverRho() { ; }
  
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (meta.Location() == QUDA_CUDA_FIELD_LOCATION) {
	cloverRhoKernel<nSpin,nColor> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
      } else {
	errorQuda("Not implemented");
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(meta.VolString(), typeid(*this).name(), aux);
    }

    long long flops() const { return 2*2*arg.clover.volumeCB*6; }
    long long bytes() const { return 2*2*arg.clover.volumeCB*(6*arg.clover.Bytes()/36); }
    void preTune() { arg.clover.save(); }
    void postTune() { arg.clover.load(); }

  };

  template <typename Float, int nSpin, int nColor, typename Clover>
  void cloverRho(Clover clover, const CloverField &meta, double rho) {
    CloverRhoArg<Float,Clover> arg(clover, rho);
    CloverRho<nSpin,nColor,CloverRhoArg<Float,Clover>> clover_rho(arg, meta);
    clover_rho.apply(0);
  }

  template <typename Float>
  void cloverRho(const CloverField &clover, double rho) {

    constexpr int nColor = 3;
    constexpr int nSpin = 4;

    if (clover.isNative()) {
      typedef typename clover_mapper<Float>::type C;
      cloverRho<Float,nSpin,nColor>(C(clover, false), clover, rho);
    } else {
      errorQuda("Clover field %d order not supported", clover.Order());
    }

  }

#endif

  void cloverRho(CloverField &clover, double rho) {

#ifdef GPU_CLOVER_DIRAC
    if (clover.Precision() == QUDA_DOUBLE_PRECISION) {
      cloverRho<double>(clover, rho);
    } else if (clover.Precision() == QUDA_SINGLE_PRECISION) {
      cloverRho<float>(clover, rho);
    } else {
      errorQuda("Precision %d not supported", clover.Precision());
    }
#else
    errorQuda("Clover has not been built");
#endif
  }

} // namespace quda
