#include "hip/hip_runtime.h"
#include <transfer.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <clover_field_order.h>
#include <complex_quda.h>
#include <index_helper.cuh>
#include <gamma.cuh>
#include <blas_cublas.h>
#include <coarse_op.cuh>

namespace quda {

  template <typename Float, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateY(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv, ColorSpinorField &av, const Transfer &T,
		  const GaugeField &g, const CloverField &c, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {

    QudaFieldLocation location = Y.Location();

    if (location == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;
      constexpr QudaCloverFieldOrder clOrder = QUDA_PACKED_CLOVER_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());
      if (c.Order() != clOrder && c.Bytes()) errorQuda("Unsupported field order %d\n", c.Order());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
      typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;
      typedef typename clover::FieldOrder<Float,fineColor,fineSpin,clOrder> cFine;

      const ColorSpinorField &v = T.Vectors(g.Location());

      F vAccessor(const_cast<ColorSpinorField&>(v));
      F uvAccessor(const_cast<ColorSpinorField&>(uv));
      F avAccessor(const_cast<ColorSpinorField&>(av));
      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarse xInvAccessor(const_cast<GaugeField&>(Xinv));
      cFine cAccessor(const_cast<CloverField&>(c), false);
      cFine cInvAccessor(const_cast<CloverField&>(c), true);

      calculateY<false,Float,fineSpin,fineColor,coarseSpin,coarseColor,gOrder>
	(yAccessor, xAccessor, xInvAccessor, uvAccessor, avAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor, Y, X, Xinv, Yhat, av, v, kappa, mu, mu_factor, dirac, matpc);

    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;
      constexpr QudaCloverFieldOrder clOrder = QUDA_FLOAT4_CLOVER_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());
      if (c.Order() != clOrder && c.Bytes()) errorQuda("Unsupported field order %d\n", c.Order());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
      typedef typename gauge::FieldOrder<Float,fineColor,1,gOrder> gFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;
      typedef typename clover::FieldOrder<Float,fineColor,fineSpin,clOrder> cFine;

      const ColorSpinorField &v = T.Vectors(g.Location());

      F vAccessor(const_cast<ColorSpinorField&>(v));
      F uvAccessor(const_cast<ColorSpinorField&>(uv));
      F avAccessor(const_cast<ColorSpinorField&>(av));
      gFine gAccessor(const_cast<GaugeField&>(g));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarse xInvAccessor(const_cast<GaugeField&>(Xinv));
      cFine cAccessor(const_cast<CloverField&>(c), false);
      cFine cInvAccessor(const_cast<CloverField&>(c), true);

      calculateY<false,Float,fineSpin,fineColor,coarseSpin,coarseColor,gOrder>
	(yAccessor, xAccessor, xInvAccessor, uvAccessor, avAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor, Y, X, Xinv, Yhat, av, v, kappa, mu, mu_factor, dirac, matpc);

    }

  }

  // template on the number of coarse degrees of freedom
  template <typename Float, int fineColor, int fineSpin>
  void calculateY(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv, ColorSpinorField &av, const Transfer &T,
		  const GaugeField &g, const CloverField &c, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) {
      calculateY<Float,fineColor,fineSpin,2,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
#if 0
    } else if (coarseCoor == 4) {
      calculateY<Float,fineColor,fineSpin,4,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 8) {
      calculateY<Float,fineColor,fineSpin,8,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 12) {
      calculateY<Float,fineColor,fineSpin,12,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 16) {
      calculateY<Float,fineColor,fineSpin,16,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 20) {
      calculateY<Float,fineColor,fineSpin,20,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
#endif
    } else if (coarseColor == 24) {
      calculateY<Float,fineColor,fineSpin,24,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
#if 0
    } else if (coarseColor == 32) {
      calculateY<Float,fineColor,fineSpin,32,coarseSpin>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
#endif
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, int fineColor>
  void calculateY(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv, ColorSpinorField &av, const Transfer &T,
		  const GaugeField &g, const CloverField &c, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (uv.Nspin() == 4) {
      calculateY<Float,fineColor,4>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported number of spins %d\n", uv.Nspin());
    }
  }

  // template on fine colors
  template <typename Float>
  void calculateY(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv, ColorSpinorField &av, const Transfer &T,
		  const GaugeField &g, const CloverField &c, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (g.Ncolor() == 3) {
      calculateY<Float,3>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateY(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv, ColorSpinorField &av, const Transfer &T,
		  const GaugeField &g, const CloverField &c, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    checkPrecision(X, Y, uv, T.Vectors(), g);

    printfQuda("Computing Y field......\n");

    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      calculateY<double>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateY<float>(Y, X, Xinv, Yhat, uv, av, T, g, c, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    printfQuda("....done computing Y field\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseOp(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, const Transfer &T,
		const cudaGaugeField &gauge, const cudaCloverField *clover,
		double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {

    QudaPrecision precision = Y.Precision();
    QudaFieldLocation location = checkLocation(Y, X, Xinv, Yhat);

    GaugeField *U = location == QUDA_CUDA_FIELD_LOCATION ? const_cast<cudaGaugeField*>(&gauge) : nullptr;
    CloverField *C = location == QUDA_CUDA_FIELD_LOCATION ? const_cast<cudaCloverField*>(clover) : nullptr;

    if (location == QUDA_CPU_FIELD_LOCATION) {
      //First make a cpu gauge field from the cuda gauge field
      int pad = 0;
      GaugeFieldParam gf_param(gauge.X(), precision, QUDA_RECONSTRUCT_NO, pad, gauge.Geometry());
      gf_param.order = QUDA_QDP_GAUGE_ORDER;
      gf_param.fixed = gauge.GaugeFixed();
      gf_param.link_type = gauge.LinkType();
      gf_param.t_boundary = gauge.TBoundary();
      gf_param.anisotropy = gauge.Anisotropy();
      gf_param.gauge = NULL;
      gf_param.create = QUDA_NULL_FIELD_CREATE;
      gf_param.siteSubset = QUDA_FULL_SITE_SUBSET;
      gf_param.nFace = 1;
      gf_param.ghostExchange = QUDA_GHOST_EXCHANGE_PAD;

      U = new cpuGaugeField(gf_param);

      //Copy the cuda gauge field to the cpu
      gauge.saveCPUField(*static_cast<cpuGaugeField*>(U));
    } else if (location == QUDA_CUDA_FIELD_LOCATION && gauge.Reconstruct() != QUDA_RECONSTRUCT_NO) {
      //Create a copy of the gauge field with no reconstruction, required for fine-grained access
      GaugeFieldParam gf_param(gauge);
      gf_param.reconstruct = QUDA_RECONSTRUCT_NO;
      gf_param.setPrecision(gf_param.precision);
      U = new cudaGaugeField(gf_param);

      U->copy(gauge);
    }

    CloverFieldParam cf_param;
    cf_param.nDim = 4;
    cf_param.pad = 0;
    cf_param.precision = clover ? clover->Precision() : QUDA_INVALID_PRECISION;

    // if we have no clover term then create an empty clover field
    for(int i = 0; i < cf_param.nDim; i++) cf_param.x[i] = clover ? clover->X()[i] : 0;

    cf_param.direct = true;
    cf_param.inverse = true;
    cf_param.clover = NULL;
    cf_param.norm = 0;
    cf_param.cloverInv = NULL;
    cf_param.invNorm = 0;
    cf_param.create = QUDA_NULL_FIELD_CREATE;
    cf_param.siteSubset = QUDA_FULL_SITE_SUBSET;

    if (location == QUDA_CUDA_FIELD_LOCATION && !clover) {
      // create a dummy cudaCloverField if one is not defined
      cf_param.order = QUDA_INVALID_CLOVER_ORDER;
      C = new cudaCloverField(cf_param);
    } else if (location == QUDA_CPU_FIELD_LOCATION) {
      //Create a cpuCloverField from the cudaCloverField
      cf_param.order = QUDA_PACKED_CLOVER_ORDER;
      C = new cpuCloverField(cf_param);
      if (clover) clover->saveCPUField(*static_cast<cpuCloverField*>(C));
    }

    //Create a field UV which holds U*V.  Has the same structure as V.
    ColorSpinorParam UVparam(T.Vectors(location));
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.location = location;

    ColorSpinorField *uv = ColorSpinorField::Create(UVparam);

    // if we are coarsening a preconditioned clover or twisted-mass operator we need
    // an additional vector to store the cloverInv * V field, else just alias v
    ColorSpinorField *av = ((matpc != QUDA_MATPC_INVALID && clover) || (dirac == QUDA_TWISTED_MASSPC_DIRAC)) ? ColorSpinorField::Create(UVparam) :
      &const_cast<ColorSpinorField&>(T.Vectors(location));

    calculateY(Y, X, Xinv, Yhat, *uv, *av, T, *U, *C, kappa, mu, mu_factor, dirac, matpc);

    if (&T.Vectors(location) != av) delete av;
    delete uv;

    if (C != clover) delete C;
    if (U != &gauge) delete U;
  }

} //namespace quda
