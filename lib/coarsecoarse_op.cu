#include <transfer.h>
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <gauge_field.h>
#include <gauge_field_order.h>
#include <complex_quda.h>
#include <index_helper.cuh>
#include <gamma.cuh>
#include <blas_cublas.h>
#include <coarse_op.cuh>

namespace quda {

  template <typename Float, int fineColor, int fineSpin, int coarseColor, int coarseSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {

    if (Y.Location() == QUDA_CPU_FIELD_LOCATION) {

      constexpr QudaFieldOrder csOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_QDP_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
      typedef typename colorspinor::FieldOrderCB<Float,2*fineSpin,fineColor,coarseColor,csOrder> F2;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder> gFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      F vAccessor(const_cast<ColorSpinorField&>(v));
      F2 uvAccessor(const_cast<ColorSpinorField&>(uv));
      gFine gAccessor(const_cast<GaugeField&>(g));
      gFine cAccessor(const_cast<GaugeField&>(clover));
      gFine cInvAccessor(const_cast<GaugeField&>(cloverInv));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarse xInvAccessor(const_cast<GaugeField&>(Xinv));

      calculateY<true,Float,fineSpin,fineColor,coarseSpin,coarseColor,gOrder>
	(yAccessor, xAccessor, xInvAccessor, uvAccessor, vAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor,
	 Y, X, Xinv, Yhat, const_cast<ColorSpinorField&>(v), v, kappa, mu, mu_factor, dirac, matpc);

    } else {

      constexpr QudaFieldOrder csOrder = QUDA_FLOAT2_FIELD_ORDER;
      constexpr QudaGaugeFieldOrder gOrder = QUDA_FLOAT2_GAUGE_ORDER;

      if (T.Vectors(Y.Location()).FieldOrder() != csOrder)
	errorQuda("Unsupported field order %d\n", T.Vectors(Y.Location()).FieldOrder());
      if (g.FieldOrder() != gOrder) errorQuda("Unsupported field order %d\n", g.FieldOrder());

      typedef typename colorspinor::FieldOrderCB<Float,fineSpin,fineColor,coarseColor,csOrder> F;
      typedef typename colorspinor::FieldOrderCB<Float,2*fineSpin,fineColor,coarseColor,csOrder> F2;
      typedef typename gauge::FieldOrder<Float,fineColor*fineSpin,fineSpin,gOrder> gFine;
      typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> gCoarse;

      const ColorSpinorField &v = T.Vectors(Y.Location());

      F vAccessor(const_cast<ColorSpinorField&>(v));
      F2 uvAccessor(const_cast<ColorSpinorField&>(uv));
      gFine gAccessor(const_cast<GaugeField&>(g));
      gFine cAccessor(const_cast<GaugeField&>(clover));
      gFine cInvAccessor(const_cast<GaugeField&>(cloverInv));
      gCoarse yAccessor(const_cast<GaugeField&>(Y));
      gCoarse xAccessor(const_cast<GaugeField&>(X));
      gCoarse xInvAccessor(const_cast<GaugeField&>(Xinv));

      calculateY<true,Float,fineSpin,fineColor,coarseSpin,coarseColor,gOrder>
	(yAccessor, xAccessor, xInvAccessor, uvAccessor, vAccessor, vAccessor, gAccessor, cAccessor, cInvAccessor,
	 Y, X, Xinv, Yhat, const_cast<ColorSpinorField&>(v), v, kappa, mu, mu_factor, dirac, matpc);

    }

  }

  // template on the number of coarse degrees of freedom
  template <typename Float, int fineColor, int fineSpin>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().Nspin()/T.Spin_bs() != 2) 
      errorQuda("Unsupported number of coarse spins %d\n",T.Vectors().Nspin()/T.Spin_bs());
    const int coarseSpin = 2;
    const int coarseColor = Y.Ncolor() / coarseSpin;

    if (coarseColor == 2) { 
      calculateYcoarse<Float,fineColor,fineSpin,2,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
#if 0
    } else if (coarseColor == 8) {
      calculateYcoarse<Float,fineColor,fineSpin,8,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 16) {
      calculateYcoarse<Float,fineColor,fineSpin,16,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
#endif
    } else if (coarseColor == 24) {
      calculateYcoarse<Float,fineColor,fineSpin,24,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else if (coarseColor == 32) {
      calculateYcoarse<Float,fineColor,fineSpin,32,coarseSpin>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // template on fine spin
  template <typename Float, int fineColor>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (T.Vectors().Nspin() == 2) {
      calculateYcoarse<Float,fineColor,2>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported number of spins %d\n", T.Vectors().Nspin());
    }
  }

  // template on fine colors
  template <typename Float>
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat,
			ColorSpinorField &uv, const Transfer &T, const GaugeField &g, const GaugeField &clover,
			const GaugeField &cloverInv, double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    if (g.Ncolor()/T.Vectors().Nspin() == 2) {
      calculateYcoarse<Float,2>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
#if 0
    } else if (g.Ncolor()/T.Vectors().Nspin() == 8) {
      calculateYcoarse<Float,8>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 16) {
      calculateYcoarse<Float,16>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
#endif
    } else if (g.Ncolor()/T.Vectors().Nspin() == 24) {
      calculateYcoarse<Float,24>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else if (g.Ncolor()/T.Vectors().Nspin() == 32) {
      calculateYcoarse<Float,32>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported number of colors %d\n", g.Ncolor());
    }
  }

  //Does the heavy lifting of creating the coarse color matrices Y
  void calculateYcoarse(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, ColorSpinorField &uv,
			const Transfer &T, const GaugeField &g, const GaugeField &clover, const GaugeField &cloverInv,
			double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {
    checkPrecision(X, Y, uv, T.Vectors(), g);

    printfQuda("Computing Y field......\n");
    if (Y.Precision() == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      calculateYcoarse<double>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (Y.Precision() == QUDA_SINGLE_PRECISION) {
      calculateYcoarse<float>(Y, X, Xinv, Yhat, uv, T, g, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);
    } else {
      errorQuda("Unsupported precision %d\n", Y.Precision());
    }
    printfQuda("....done computing Y field\n");
  }

  //Calculates the coarse color matrix and puts the result in Y.
  //N.B. Assumes Y, X have been allocated.
  void CoarseCoarseOp(GaugeField &Y, GaugeField &X, GaugeField &Xinv, GaugeField &Yhat, const Transfer &T,
		      const GaugeField &gauge, const GaugeField &clover, const GaugeField &cloverInv,
		      double kappa, double mu, double mu_factor, QudaDiracType dirac, QudaMatPCType matpc) {

    QudaPrecision precision = Y.Precision();
    QudaFieldLocation location = checkLocation(X, Y, Xinv, Yhat, gauge, clover, cloverInv);

    //Create a field UV which holds U*V.  Has the same similar
    //structure to V but double the number of spins so we can store
    //the four distinct block chiral multiplications in a single UV
    //computation.
    ColorSpinorParam UVparam(T.Vectors(location));
    UVparam.create = QUDA_ZERO_FIELD_CREATE;
    UVparam.location = location;
    UVparam.nSpin *= 2; // so nSpin == 4

    ColorSpinorField *uv = ColorSpinorField::Create(UVparam);

    calculateYcoarse(Y, X, Xinv, Yhat, *uv, T, gauge, clover, cloverInv, kappa, mu, mu_factor, dirac, matpc);

    delete uv;
  }
  
} //namespace quda
