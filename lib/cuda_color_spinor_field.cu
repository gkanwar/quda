#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

int zeroCopy = 0;

namespace quda {

  bool cudaColorSpinorField::initGhostFaceBuffer = false;
  size_t cudaColorSpinorField::ghostFaceBytes = 0;

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      errorQuda("not implemented");
    }
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false),
    ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) || 
         (param.create == QUDA_REFERENCE_FIELD_CREATE && (param.is_composite || param.is_component))) {
      reset(param);
    } else {
      // errorQuda("Undefined behaviour"); // else silent bug possible?
      reset(param);
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }

      if (composite_descr.is_component && !(src.SiteSubset() == QUDA_FULL_SITE_SUBSET && this->SiteSubset() == QUDA_PARITY_SITE_SUBSET)) 
      {//setup eigenvector form the set
        v    = (void*)((char*)v    + composite_descr.id*bytes);         
        norm = (void*)((char*)norm + composite_descr.id*norm_bytes);         
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false),
      ghostTexInit(false), ghost_field_tex{nullptr,nullptr}, bufferMessageHandler(0)
  {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	destroyComms(); // not sure if this necessary
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroyComms();
    destroy();
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      switch(mem_type) {
      case QUDA_MEMORY_DEVICE:
	v = pool_device_malloc(bytes);
	if (precision == QUDA_HALF_PRECISION) norm = pool_device_malloc(norm_bytes);
	break;
      case QUDA_MEMORY_MAPPED:
	v_h = mapped_malloc(bytes);
	hipHostGetDevicePointer(&v, v_h, 0); // set the matching device pointer
	if (precision == QUDA_HALF_PRECISION) {
	  norm_h = mapped_malloc(norm_bytes);
	  hipHostGetDevicePointer(&norm, norm_h, 0); // set the matching device pointer
	}
	break;
      default:
	errorQuda("Unsupported memory type %d", mem_type);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      if(composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) {
	if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
	  
        ColorSpinorParam param;
        param.siteSubset = QUDA_FULL_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite   = false;
        param.composite_dim  = 0;
        param.is_component = true;
	param.mem_type = mem_type;

        components.reserve(composite_descr.dim);
        for(int cid = 0; cid < composite_descr.dim; cid++) {
	  param.component_id = cid;
	  components.push_back(new cudaColorSpinorField(*this, param));
        }
      } else {
        // create the associated even and odd subsets
        ColorSpinorParam param;
        param.siteSubset = QUDA_PARITY_SITE_SUBSET;
        param.nDim = nDim;
        memcpy(param.x, x, nDim*sizeof(int));
        param.x[0] /= 2; // set single parity dimensions
        param.create = QUDA_REFERENCE_FIELD_CREATE;
        param.v = v;
        param.norm = norm;
        param.is_composite  = false;
        param.composite_dim = 0;
        param.is_component  = composite_descr.is_component;
        param.component_id  = composite_descr.id;
	param.mem_type = mem_type;

        even = new cudaColorSpinorField(*this, param);
        odd = new cudaColorSpinorField(*this, param);

        // need this hackery for the moment (need to locate the odd pointers half way into the full field)
        (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
        if (precision == QUDA_HALF_PRECISION) 
	  (dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

#ifdef USE_TEXTURE_OBJECTS
        dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
        dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
      }
    } else { //siteSubset == QUDA_PARITY_SITE_SUBSET

      //! setup an object for selected eigenvector (the 1st one as a default):
      if (composite_descr.is_composite && (create != QUDA_REFERENCE_FIELD_CREATE)) 
      {
         if(composite_descr.dim <= 0) errorQuda("\nComposite size is not defined\n");
         //if(bytes > 1811939328) warningQuda("\nCUDA API probably won't be able to create texture object for the eigenvector set... Object size is : %u bytes\n", bytes);
         if (getVerbosity() == QUDA_DEBUG_VERBOSE) printfQuda("\nEigenvector set constructor...\n");
         // create the associated even and odd subsets
         ColorSpinorParam param;
         param.siteSubset = QUDA_PARITY_SITE_SUBSET;
         param.nDim = nDim;
         memcpy(param.x, x, nDim*sizeof(int));
         param.x[4] = 1;
         param.create = QUDA_REFERENCE_FIELD_CREATE;
         param.v = v;
         param.norm = norm;
         param.is_composite   = false;
         param.composite_dim  = 0;
         param.is_component = true;
	 param.mem_type = mem_type;

         //reserve eigvector set
         components.reserve(composite_descr.dim);
         //setup volume, [real_]length and stride for a single eigenvector
         for(int cid = 0; cid < composite_descr.dim; cid++)
         {
            param.component_id = cid;
            components.push_back(new cudaColorSpinorField(*this, param));

#ifdef USE_TEXTURE_OBJECTS //(a lot of texture objects...)
            dynamic_cast<cudaColorSpinorField*>(components[cid])->destroyTexObject();
            dynamic_cast<cudaColorSpinorField*>(components[cid])->createTexObject();
#endif
         }
      }
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else if(!composite_descr.is_composite) {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      } else { //temporary hack for the full spinor field sets, manual zeroPad for each component:
	for(int cid = 0; cid < composite_descr.dim; cid++) {
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Even())).zeroPad();
	  (dynamic_cast<cudaColorSpinorField&>(components[cid]->Odd())).zeroPad();
	}
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    // if (!composite_descr.is_composite || composite_descr.is_component)
      createTexObject();
#endif
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered and coarse fields in half and single are always two component
      if ( (nSpin == 1 || nSpin == 2) && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component (double2 is spread across int4)
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);

      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);
      }
      
      texInit = true;

      checkCudaError();
    }
  }

  void cudaColorSpinorField::createGhostTexObject() const {
    // create the ghost texture object
    if (isNative() && ghost_bytes) {
      if (ghostTexInit) errorQuda("Already bound ghost texture");

      for (int b=0; b<2; b++) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
	else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

	// staggered and coarse fields in half and single are always two component
	if ( (nSpin == 1 || nSpin == 2) && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	  desc.x = 8*precision;
	  desc.y = 8*precision;
	  desc.z = 0;
	  desc.w = 0;
	} else { // all others are four component (double2 is spread across int4)
	  desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	  desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	}

	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = ghost_recv_buffer_d[b];
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = ghost_bytes;

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
	else texDesc.readMode = hipReadModeElementType;

	hipCreateTextureObject(&ghostTex[b], &resDesc, &texDesc, NULL);

	// second set of ghost texture map to the host-mapped pinned receive buffers
	resDesc.res.linear.devPtr = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
	hipCreateTextureObject(&ghostTex[2+b], &resDesc, &texDesc, NULL);

	if (precision == QUDA_HALF_PRECISION) {
	  hipChannelFormatDesc desc;
	  memset(&desc, 0, sizeof(hipChannelFormatDesc));
	  desc.f = hipChannelFormatKindFloat;
	  desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;

	  hipResourceDesc resDesc;
	  memset(&resDesc, 0, sizeof(resDesc));
	  resDesc.resType = hipResourceTypeLinear;
	  resDesc.res.linear.devPtr = ghost_recv_buffer_d[b];
	  resDesc.res.linear.desc = desc;
	  resDesc.res.linear.sizeInBytes = ghost_bytes;

	  hipTextureDesc texDesc;
	  memset(&texDesc, 0, sizeof(texDesc));
	  texDesc.readMode = hipReadModeElementType;

	  hipCreateTextureObject(&ghostTexNorm[b], &resDesc, &texDesc, NULL);

	  resDesc.res.linear.devPtr = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
	  hipCreateTextureObject(&ghostTexNorm[2+b], &resDesc, &texDesc, NULL);
	}

	ghost_field_tex[b] = ghost_recv_buffer_d[b];
	ghost_field_tex[2+b] = static_cast<char*>(ghost_pinned_buffer_hd[b])+ghost_bytes;
      } // buffer index

      ghostTexInit = true;

      checkCudaError();
    }

  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if (ghost_bytes) {
	for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTex[i]);
      }
      if (precision == QUDA_HALF_PRECISION) {
        hipDestroyTextureObject(texNorm);
        if (ghost_bytes) {
	  for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTexNorm[i]);
	}
      }
      texInit = false;
    }
  }

  void cudaColorSpinorField::destroyGhostTexObject() const {
    if (isNative() && ghostTexInit) {
      for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTex[i]);
      if (precision == QUDA_HALF_PRECISION) {
	for (int i=0; i<4; i++) hipDestroyTextureObject(ghostTexNorm[i]);
      }
      ghostTexInit = false;
    }
  }
#endif

  void cudaColorSpinorField::destroy() {

    if (alloc) {
      switch(mem_type) {
      case QUDA_MEMORY_DEVICE:
	pool_device_free(v);
	if (precision == QUDA_HALF_PRECISION) pool_device_free(norm);
	break;
      case QUDA_MEMORY_MAPPED:
	host_free(v_h);
	if (precision == QUDA_HALF_PRECISION) host_free(norm_h);
	break;
      default:
	errorQuda("Unsupported memory type %d", mem_type);
      }
    }


    if (composite_descr.is_composite) 
    {
       CompositeColorSpinorField::iterator vec;
       for(vec = components.begin(); vec != components.end(); vec++) delete *vec;
    } 

    if ((siteSubset == QUDA_FULL_SITE_SUBSET && !composite_descr.is_composite) || (siteSubset == QUDA_FULL_SITE_SUBSET && composite_descr.is_component)) {
      delete even;
      delete odd;
    }

#ifdef USE_TEXTURE_OBJECTS
    // if (!composite_descr.is_composite || composite_descr.is_component)
      destroyTexObject();
#endif

  }

  void cudaColorSpinorField::backup() const {
    if (backed_up) errorQuda("Gauge field already backed up");
    backup_h = new char[bytes];
    hipMemcpy(backup_h, v, bytes, hipMemcpyDeviceToHost);
    if (norm_bytes) {
      backup_norm_h = new char[norm_bytes];
      hipMemcpy(backup_norm_h, norm, norm_bytes, hipMemcpyDeviceToHost);
    }
    checkCudaError();
    backed_up = true;
  }

  void cudaColorSpinorField::restore() {
    if (!backed_up) errorQuda("Cannot restore since not backed up");
    hipMemcpy(v, backup_h, bytes, hipMemcpyHostToDevice);
    delete []backup_h;
    if (norm_bytes) {
      hipMemcpy(v, backup_norm_h, norm_bytes, hipMemcpyHostToDevice);
      delete []backup_norm_h;
    }
    checkCudaError();
    backed_up = false;
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }

  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = 0;//(stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;

    if (composite_descr.is_composite && !composite_descr.is_component){//we consider the whole eigenvector set:
      Npad      *= composite_descr.dim;
      pad_bytes /= composite_descr.dim;
    }

    size_t pitch = ((!composite_descr.is_composite || composite_descr.is_component) ? stride : composite_descr.stride)*fieldOrder*precision;
    char   *dst  = (char*)v + ((!composite_descr.is_composite || composite_descr.is_component) ? volume : composite_descr.volume)*fieldOrder*precision;
    if (pad_bytes) hipMemset2D(dst, pitch, 0, pad_bytes, Npad);

    //for (int i=0; i<Npad; i++) {
    //  if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    //}
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    if (this->GammaBasis() != src.GammaBasis()) errorQuda("cannot call this copy with different basis");
    blas::copy(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative() &&
	this->GammaBasis() == src.GammaBasis()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (reorder_location() == QUDA_CPU_FIELD_LOCATION &&typeid(src) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes + norm_bytes);
      memset(buffer, 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding

      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, buffer, 0, static_cast<char*>(buffer)+bytes, 0);

      qudaMemcpy(v, buffer, bytes, hipMemcpyHostToDevice);
      qudaMemcpy(norm, static_cast<char*>(buffer)+bytes, norm_bytes, hipMemcpyHostToDevice);

      pool_pinned_free(buffer);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src=nullptr, *srcNorm=nullptr, *buffer=nullptr;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();
	qudaMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	qudaMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	buffer = pool_pinned_malloc(src.Bytes()+src.NormBytes());
	memcpy(buffer, src.V(), src.Bytes());
	memcpy(static_cast<char*>(buffer)+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, buffer, 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);

      if (zeroCopy) pool_pinned_free(buffer);
    }

    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (reorder_location() == QUDA_CPU_FIELD_LOCATION && typeid(dest) == typeid(cpuColorSpinorField)) {
      void *buffer = pool_pinned_malloc(bytes+norm_bytes);
      qudaMemcpy(buffer, v, bytes, hipMemcpyDeviceToHost);
      qudaMemcpy(static_cast<char*>(buffer)+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 0, buffer, 0, static_cast<char*>(buffer)+bytes);
      pool_pinned_free(buffer);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst=nullptr, *dstNorm=nullptr, *buffer=nullptr;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	buffer = pool_pinned_malloc(dest.Bytes()+dest.NormBytes());
	hipHostGetDevicePointer(&dst, buffer, 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	qudaMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	qudaMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), buffer, dest.Bytes());
	memcpy(dest.Norm(), static_cast<char*>(buffer) + dest.Bytes(), dest.NormBytes());
      }

      if (zeroCopy) pool_pinned_free(buffer);
    }

    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(int nFace, bool spin_project) const {

    if (!comm_partitioned()) {
      for (int i=0; i<4; i++) ghost_face_bytes[i] = 0;
      return;
    }

    createGhostZone(nFace, spin_project);

    // temporary work around until the ghost buffer for fine and
    // coarse grid are merged: this ensures we reset the fine ghost
    // buffer if the coarse grid operator allocates a ghost buffer
    // that is larger than the fine grid operator
    static size_t ghostFaceBytes_ = 0;

    // only allocate if not already allocated or buffer required is bigger than previously
    if ( !initGhostFaceBuffer || ghost_bytes > ghostFaceBytes || ghost_bytes > ghostFaceBytes_) {

      if (initGhostFaceBuffer) {
#ifdef USE_TEXTURE_OBJECTS
	destroyGhostTexObject();
#endif
	if (ghost_bytes) {
	  for (int b=0; b<2; b++) {
	    device_pinned_free(ghost_recv_buffer_d[b]);
	    device_pinned_free(ghost_send_buffer_d[b]);
	    host_free(ghost_pinned_buffer_h[b]);
	  }
	}
      }

      if (ghost_bytes > 0) {
	for (int b=0; b<2; ++b) {
	  // gpu receive buffer (use pinned allocator to avoid this being redirected, e.g., by QDPJIT)
	  ghost_recv_buffer_d[b] = device_pinned_malloc(ghost_bytes);

	  // gpu send buffer (use pinned allocator to avoid this being redirected, e.g., by QDPJIT)
	  ghost_send_buffer_d[b] = device_pinned_malloc(ghost_bytes);

	  // pinned buffer used for sending and receiving
	  ghost_pinned_buffer_h[b] = mapped_malloc(2*ghost_bytes);

	  // set the matching device-mapper pointer
	  hipHostGetDevicePointer(&ghost_pinned_buffer_hd[b], ghost_pinned_buffer_h[b], 0);
	}

	initGhostFaceBuffer = true;
	ghostFaceBytes = ghost_bytes;
	ghostFaceBytes_ = ghost_bytes;
      }

      LatticeField::ghost_field_reset = true; // this signals that we must reset the IPC comms
    }

#ifdef USE_TEXTURE_OBJECTS
    // ghost texture is per object
    if (ghost_field_tex[0] != ghost_recv_buffer_d[0] || ghost_field_tex[1] != ghost_recv_buffer_d[1]) destroyGhostTexObject();
    if (!ghostTexInit) createGhostTexObject();
#endif
  }

  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    destroyIPCComms();

    if (!initGhostFaceBuffer) return;
  
    for (int b=0; b<2; b++) {
      // free receive buffer
      if (ghost_recv_buffer_d[b]) device_pinned_free(ghost_recv_buffer_d[b]);
      ghost_recv_buffer_d[b] = nullptr;

      // free send buffer
      if (ghost_send_buffer_d[b]) device_pinned_free(ghost_send_buffer_d[b]);
      ghost_send_buffer_d[b] = nullptr;

      // free pinned memory buffers
      if (ghost_pinned_buffer_h[b]) host_free(ghost_pinned_buffer_h[b]);
      ghost_pinned_buffer_h[b] = nullptr;
      ghost_pinned_buffer_hd[b] = nullptr;
    }
    initGhostFaceBuffer = false;
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const int nFace, const QudaParity parity, 
                                       const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream, 
				       MemoryLocation location [2*QUDA_MAX_DIM], double a, double b)
  {
#ifdef MULTI_GPU
    int face_num = (dir == QUDA_BACKWARDS) ? 0 : (dir == QUDA_FORWARDS) ? 1 : 2;
    void *packBuffer[2*QUDA_MAX_DIM];

    for (int dim=0; dim<4; dim++) {
      for (int dir=0; dir<2; dir++) {
	switch(location[2*dim+dir]) {
	case Device: // pack to local device buffer
	  packBuffer[2*dim+dir] = my_face_dim_dir_d[bufferIndex][dim][dir]; break;
	case Host:   // pack to zero-copy memory
	  packBuffer[2*dim+dir] = my_face_dim_dir_hd[bufferIndex][dim][dir]; break;
	default: errorQuda("Undefined location %d", location[2*dim+dir]);
	}
      }
    }

    packFace(packBuffer, *this, location, nFace, dagger, parity, dim, face_num, *stream, a, b);
#else
    errorQuda("packGhost not built on single-GPU build");
#endif
  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int nFace, const int dim, 
				       const QudaDirection dir, const int dagger, 
				       hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom
    int Npad = Nint / Nvec; // number Nvec buffers we have
    
    if (dim !=3 || getKernelPackT()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;

      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);

      void* gpu_buf = (dir == QUDA_BACKWARDS) ? my_face_dim_dir_d[bufferIndex][dim][0] : my_face_dim_dir_d[bufferIndex][dim][1];

      hipMemcpyAsync(ghost_spinor, gpu_buf, ghost_face_bytes[dim], hipMemcpyDeviceToHost, *stream);

    } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) { // do multiple cudaMemcpys

      const int x4 = nDim==5 ? x[4] : 1;
      const int Nt_minus1_offset = (volumeCB - nFace*ghostFace[3])/x4; // N_t -1 = Vh-Vsh

      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      size_t len = nFace*(ghostFace[3]/x4)*Nvec*precision;
      size_t dpitch = x4*len;
      size_t spitch = composite_descr.is_composite ? composite_descr.stride*Nvec*precision : stride*Nvec*precision;

      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next
      for (int s=0; s<x4; s++) { // loop over multiple 4-d volumes (if they exist)
	void *dst = (char*)ghost_spinor + s*len;
	void *src = composite_descr.is_composite ? (char*)v + (offset + s*composite_descr.volumeCB*Npad)*Nvec*precision : (char*)v + (offset + s*(volumeCB/x4))*Nvec*precision;
	hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

	if (precision == QUDA_HALF_PRECISION) {
	  size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
	  int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	  void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision + s*len;
	  void *src = (char*)norm + norm_offset + s*(volumeCB/x4)*sizeof(float);
	  hipMemcpyAsync(dst, src, len, hipMemcpyDeviceToHost, *stream);
	}
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper) {
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }


  void cudaColorSpinorField::unpackGhost(const void* ghost_spinor, const int nFace, 
					 const int dim, const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    const void *src = ghost_spinor;
    int ghost_offset = (dir == QUDA_BACKWARDS) ? ghostOffset[dim][0] : ghostOffset[dim][1];
    void *ghost_dst = (char*)ghost_recv_buffer_d[bufferIndex] + precision*ghost_offset;

    hipMemcpyAsync(ghost_dst, src, ghost_face_bytes[dim], hipMemcpyHostToDevice, *stream);
  }


  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhostExtended(const int nFace, const int R[], const QudaParity parity,
					       const int dim, const QudaDirection dir,
					       const int dagger, hipStream_t *stream, bool zero_copy)
  {
#ifdef MULTI_GPU
    int face_num = (dir == QUDA_BACKWARDS) ? 0 : (dir == QUDA_FORWARDS) ? 1 : 2;
    void *packBuffer[2*QUDA_MAX_DIM];
    MemoryLocation location[2*QUDA_MAX_DIM];

    if (zero_copy) {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_hd[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_hd[bufferIndex][d][1];
	location[2*d+0] = Host;
	location[2*d+1] = Host;
      }
    } else {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_d[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_d[bufferIndex][d][1];
	location[2*d+0] = Device;
	location[2*d+1] = Device;
      }
    }

    packFaceExtended(packBuffer, *this, location, nFace, R, dagger, parity, dim, face_num, *stream);
#else
    errorQuda("packGhostExtended not built on single-GPU build");
#endif

  }


  // copy data from host buffer into boundary region of device field
  void cudaColorSpinorField::unpackGhostExtended(const void* ghost_spinor, const int nFace, const QudaParity parity,
                                                 const int dim, const QudaDirection dir, 
                                                 const int dagger, hipStream_t* stream, bool zero_copy)
  {
    // First call the regular unpackGhost routine to copy data into the `usual' ghost-zone region 
    // of the data array 
    unpackGhost(ghost_spinor, nFace, dim, dir, dagger, stream);

    // Next step is to copy data from the ghost zone back to the interior region
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim][0];
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

#ifdef MULTI_GPU
    const int face_num = 2;
    const bool unpack = true;
    const int R[4] = {0,0,0,0};
    void *packBuffer[2*QUDA_MAX_DIM];
    MemoryLocation location[2*QUDA_MAX_DIM];

    if (zero_copy) {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_hd[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_hd[bufferIndex][d][1];
	location[2*d+0] = Host;
	location[2*d+1] = Host;
      }
    } else {
      for (int d=0; d<4; d++) {
	packBuffer[2*d+0] = my_face_dim_dir_d[bufferIndex][d][0];
	packBuffer[2*d+1] = my_face_dim_dir_d[bufferIndex][d][1];
	location[2*d+0] = Device;
	location[2*d+1] = Device;
      }
    }

    packFaceExtended(packBuffer, *this, location, nFace, R, dagger, parity, dim, face_num, *stream, unpack);
#else
    errorQuda("unpackGhostExtended not built on single-GPU build");
#endif
  }


  hipStream_t *stream;

  void cudaColorSpinorField::createComms(int nFace, bool spin_project) {

    allocateGhostBuffer(nFace,spin_project); // allocate the ghost buffer if not yet allocated

    // ascertain if this instance needs its comms buffers to be updated
    bool comms_reset = ghost_field_reset || // FIXME add send buffer check
      (my_face_h[0] != ghost_pinned_buffer_h[0]) || (my_face_h[1] != ghost_pinned_buffer_h[1]) || // pinned buffers
      (ghost_field_tex[0] != ghost_recv_buffer_d[0]) || (ghost_field_tex[1] != ghost_recv_buffer_d[1]); // receive buffers

    if (!initComms || comms_reset) {

      destroyComms(); // if we are requesting a new number of faces destroy and start over

      int Nint = nColor * nSpin * 2 / (nSpin == 4 && spin_project ? 2 : 1); // number of internal degrees of freedom

      for (int i=0; i<nDimComms; i++) { // compute size of ghost buffers required
	if (!commDimPartitioned(i)) { ghost_face_bytes[i] = 0; continue; }
	ghost_face_bytes[i] = nFace*ghostFace[i]*Nint*precision;
	if (precision == QUDA_HALF_PRECISION) ghost_face_bytes[i] += nFace*ghostFace[i]*sizeof(float);
      }

      // initialize the ghost pinned buffers
      for (int b=0; b<2; b++) {
	my_face_h[b] = ghost_pinned_buffer_h[b];
	my_face_hd[b] = ghost_pinned_buffer_hd[b];
	from_face_h[b] = static_cast<char*>(my_face_h[b]) + ghost_bytes;
	from_face_hd[b] = static_cast<char*>(my_face_hd[b]) + ghost_bytes;
      }

      // initialize the ghost receive pointers
      for (int i=0; i<nDimComms; ++i) {
	if (commDimPartitioned(i)) {
	  for (int b=0; b<2; b++) {
	    ghost[b][i] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostOffset[i][0]*precision;
	    if (precision == QUDA_HALF_PRECISION)
	      ghostNorm[b][i] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostNormOffset[i][0]*QUDA_SINGLE_PRECISION;
	  }
	}
      }

      // initialize ghost send pointers
      size_t offset = 0;
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;

	for (int b=0; b<2; ++b) {
	  my_face_dim_dir_h[b][i][0] = static_cast<char*>(my_face_h[b]) + offset;
	  from_face_dim_dir_h[b][i][0] = static_cast<char*>(from_face_h[b]) + offset;

	  my_face_dim_dir_hd[b][i][0] = static_cast<char*>(my_face_hd[b]) + offset;
	  from_face_dim_dir_hd[b][i][0] = static_cast<char*>(from_face_hd[b]) + offset;

	  my_face_dim_dir_d[b][i][0] = static_cast<char*>(ghost_send_buffer_d[b]) + offset;
	  from_face_dim_dir_d[b][i][0] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostOffset[i][0]*precision;
	} // loop over b
	offset += ghost_face_bytes[i];

	for (int b=0; b<2; ++b) {
	  my_face_dim_dir_h[b][i][1] = static_cast<char*>(my_face_h[b]) + offset;
	  from_face_dim_dir_h[b][i][1] = static_cast<char*>(from_face_h[b]) + offset;

	  my_face_dim_dir_hd[b][i][1] = static_cast<char*>(my_face_hd[b]) + offset;
	  from_face_dim_dir_hd[b][i][1] = static_cast<char*>(from_face_hd[b]) + offset;

	  my_face_dim_dir_d[b][i][1] = static_cast<char*>(ghost_send_buffer_d[b]) + offset;
	  from_face_dim_dir_d[b][i][1] = static_cast<char*>(ghost_recv_buffer_d[b]) + ghostOffset[i][1]*precision;
	} // loop over b
	offset += ghost_face_bytes[i];

      } // loop over dimension

      bool gdr = comm_gdr_enabled(); // only allocate rdma buffers if GDR enabled

      // initialize the message handlers
      for (int i=0; i<nDimComms; i++) {
	if (!commDimPartitioned(i)) continue;

	for (int b=0; b<2; ++b) {
	  mh_send_fwd[b][i] = comm_declare_send_relative(my_face_dim_dir_h[b][i][1], i, +1, ghost_face_bytes[i]);
	  mh_send_back[b][i] = comm_declare_send_relative(my_face_dim_dir_h[b][i][0], i, -1, ghost_face_bytes[i]);

	  mh_recv_fwd[b][i] = comm_declare_receive_relative(from_face_dim_dir_h[b][i][1], i, +1, ghost_face_bytes[i]);
	  mh_recv_back[b][i] = comm_declare_receive_relative(from_face_dim_dir_h[b][i][0], i, -1, ghost_face_bytes[i]);

	  mh_send_rdma_fwd[b][i] = gdr ? comm_declare_send_relative(my_face_dim_dir_d[b][i][1], i, +1, ghost_face_bytes[i]) : nullptr;
	  mh_send_rdma_back[b][i] = gdr ? comm_declare_send_relative(my_face_dim_dir_d[b][i][0], i, -1, ghost_face_bytes[i]) : nullptr;

	  mh_recv_rdma_fwd[b][i] = gdr ? comm_declare_receive_relative(from_face_dim_dir_d[b][i][1], i, +1, ghost_face_bytes[i]) : nullptr;
	  mh_recv_rdma_back[b][i] = gdr ? comm_declare_receive_relative(from_face_dim_dir_d[b][i][0], i, -1, ghost_face_bytes[i]) : nullptr;
	} // loop over b

      } // loop over dimension
     
      initComms = true;
      checkCudaError();
    }

    if (LatticeField::ghost_field_reset) destroyIPCComms();
    createIPCComms();
  }

  void cudaColorSpinorField::destroyComms()
  {
    if (initComms) {

      for (int b=0; b<2; ++b) {
	for (int i=0; i<nDimComms; i++) {
	  if (commDimPartitioned(i)) {
	    if (mh_recv_fwd[b][i]) comm_free(mh_recv_fwd[b][i]);
	    if (mh_recv_back[b][i]) comm_free(mh_recv_back[b][i]);
	    if (mh_send_fwd[b][i]) comm_free(mh_send_fwd[b][i]);
	    if (mh_send_back[b][i]) comm_free(mh_send_back[b][i]);

	    if (mh_recv_rdma_fwd[b][i]) comm_free(mh_recv_rdma_fwd[b][i]);
	    if (mh_recv_rdma_back[b][i]) comm_free(mh_recv_rdma_back[b][i]);
	    if (mh_send_rdma_fwd[b][i]) comm_free(mh_send_rdma_fwd[b][i]);
	    if (mh_send_rdma_back[b][i]) comm_free(mh_send_rdma_back[b][i]);
	  }
	}
      } // loop over b

      initComms = false;
      checkCudaError();
    }

  }

  void cudaColorSpinorField::streamInit(hipStream_t *stream_p) {
    stream = stream_p;
  }

  void cudaColorSpinorField::pack(int nFace, int parity, int dagger, int stream_idx,
				  MemoryLocation location[2*QUDA_MAX_DIM], double a, double b)
  {
    createComms(nFace); // must call this first

    const int dim=-1; // pack all partitioned dimensions
 
    packGhost(nFace, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[stream_idx], location, a, b);
  }

  void cudaColorSpinorField::packExtended(const int nFace, const int R[], const int parity, 
                                          const int dagger, const int dim,
                                          hipStream_t *stream_p, const bool zero_copy)
  {
    createComms(nFace); // must call this first

    stream = stream_p;
 
    packGhostExtended(nFace, R, (QudaParity)parity, dim, QUDA_BOTH_DIRS, dagger, &stream[zero_copy ? 0 : (Nstream-1)], zero_copy);
  }

  void cudaColorSpinorField::gather(int nFace, int dagger, int dir, hipStream_t* stream_p)
  {
    int dim = dir/2;

    // If stream_p != 0, use pack_stream, else use the stream array
    hipStream_t *pack_stream = (stream_p) ? stream_p : stream+dir;

    if (dir%2 == 0) {
      // backwards copy to host
      if (comm_peer2peer_enabled(0,dim)) return;

      sendGhost(my_face_dim_dir_h[bufferIndex][dim][0], nFace, dim, QUDA_BACKWARDS, dagger, pack_stream);
    } else {
      // forwards copy to host
      if (comm_peer2peer_enabled(1,dim)) return;

      sendGhost(my_face_dim_dir_h[bufferIndex][dim][1], nFace, dim, QUDA_FORWARDS, dagger, pack_stream);
    }
  }


  void cudaColorSpinorField::recvStart(int nFace, int dir, int dagger, hipStream_t* stream_p, bool gdr) {

    int dim = dir/2;
    if (!commDimPartitioned(dim)) return;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but not GDR is not enabled");

    if (dir%2 == 0) { // sending backwards
      if (comm_peer2peer_enabled(1,dim)) {
	// receive from the processor in the +1 direction
	comm_start(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
        // Prepost receive
        comm_start(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
        // Prepost receive
        comm_start(mh_recv_fwd[bufferIndex][dim]);
      }
    } else { //sending forwards
      // Prepost receive
      if (comm_peer2peer_enabled(0,dim)) {
	comm_start(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
        comm_start(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
        comm_start(mh_recv_back[bufferIndex][dim]);
      }
    }
  }


  void cudaColorSpinorField::sendStart(int nFace, int d, int dagger, hipStream_t* stream_p, bool gdr) {

    int dim = d/2;
    int dir = d%2;
    if (!commDimPartitioned(dim)) return;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but not GDR is not enabled");

    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int Nint = (nColor * nSpin * 2)/(nSpin == 4 ? 2 : 1); // (spin proj.) degrees of freedom
    int Npad = Nint/Nvec;

    if (!comm_peer2peer_enabled(dir,dim)) {
      if (dir == 0)
	if (gdr) comm_start(mh_send_rdma_back[bufferIndex][dim]);
	else comm_start(mh_send_back[bufferIndex][dim]);
      else
	if (gdr) comm_start(mh_send_rdma_fwd[bufferIndex][dim]);
	else comm_start(mh_send_fwd[bufferIndex][dim]);
    } else { // doing peer-to-peer
      hipStream_t *copy_stream = (stream_p) ? stream_p : stream + d;

      // all goes here
      void* ghost_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
	+ precision*ghostOffset[dim][(dir+1)%2];
      void *ghost_norm_dst = static_cast<char*>(ghost_remote_send_buffer_d[bufferIndex][dim][dir])
	+ QUDA_SINGLE_PRECISION*ghostNormOffset[dim][(d+1)%2];

      if (dim != 3 || getKernelPackT()) {

	hipMemcpyAsync(ghost_dst,
			my_face_dim_dir_d[bufferIndex][dim][dir],
			ghost_face_bytes[dim],
			hipMemcpyDeviceToDevice,
			*copy_stream); // copy to forward processor

      } else if (this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET) {

	const int x4 = nDim==5 ? x[4] : 1;
	const int Nt_minus_offset = (volumeCB - nFace*ghostFace[3])/x4;

	int offset = 0;
	if (nSpin == 1) {
	  offset = (dir == 0) ? 0 : Nt_minus_offset;
	} else if (nSpin == 4) {
	  // !dagger: send lower components backwards, send upper components forwards
	  // dagger: send upper components backwards, send lower components forwards
	  bool upper = dagger ? true : false;
	  if (dir == 1) upper = !upper;
	  int lower_spin_offset = Npad*stride;
	  if (dir == 0) {
	    offset = upper ? 0 : lower_spin_offset;
	  } else {
	    offset = (upper) ? Nt_minus_offset : lower_spin_offset + Nt_minus_offset;
	  }
	}

	size_t len = nFace*(ghostFace[3]/x4)*Nvec*precision;
	size_t dpitch = x4*len;
        size_t spitch = composite_descr.is_composite ? composite_descr.stride*Nvec*precision : stride*Nvec*precision;

	for (int s=0; s<x4; s++) {
	  void *dst = (char*)ghost_dst + s*len;
	  void *src = composite_descr.is_composite ? (char*)v + (offset + s*composite_descr.volumeCB*Npad)*Nvec*precision : (char*)v + (offset + s*(volumeCB/x4))*Nvec*precision;
	  // start the copy
	  hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	  if (precision == QUDA_HALF_PRECISION) {
	    size_t len = nFace*(ghostFace[3]/x4)*sizeof(float);
	    int norm_offset = (dir == 0) ? 0 : Nt_minus_offset*sizeof(float);
	    void *dst = (char*)ghost_norm_dst + s*len;
	    void *src = static_cast<char*>(norm) + norm_offset + s*(volumeCB/x4)*sizeof(float);
	    hipMemcpyAsync(dst, src, len, hipMemcpyDeviceToDevice, *copy_stream);
	  }
	}
      } else { // twisted doublet
	int flavorVolume = volume / 2;
	int flavorTFace  = ghostFace[3] / 2;
	int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
	int flavor2_Nt_minus1_offset = (volume - flavorTFace);
	int flavor1_offset = 0;
	int flavor2_offset = 0;
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back
	if (dir == 1) upper = !upper;
	int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
	if (upper) {
	  flavor1_offset = (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
	  flavor2_offset = (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
	}else{
	  flavor1_offset = lower_spin_offset + (dir == 0 ? 0 : flavor1_Nt_minus1_offset);
	  flavor2_offset = lower_spin_offset + (dir == 0 ? flavorVolume : flavor2_Nt_minus1_offset);
	}

	// QUDA Memcpy NPad's worth.
	//  -- Dest will point to the right beginning PAD.
	//  -- Each Pad has size Nvec*Vsh Floats.
	//  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

	void *src = static_cast<char*>(v) + flavor1_offset*Nvec*precision;
	size_t len = flavorTFace*Nvec*precision;
	size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
	size_t dpitch = 2*len;
	hipMemcpy2DAsync(ghost_dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	src = static_cast<char*>(v) + flavor2_offset*Nvec*precision;
	hipMemcpy2DAsync(static_cast<char*>(ghost_dst)+len, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToDevice, *copy_stream);

	if (precision == QUDA_HALF_PRECISION) {
	  int norm_offset = (dir == 0) ? 0 : flavor1_Nt_minus1_offset*sizeof(float);
	  void *src = static_cast<char*>(norm) + norm_offset;
	  size_t dpitch = flavorTFace*sizeof(float);
	  size_t spitch = flavorVolume*sizeof(float);
	  hipMemcpy2DAsync(ghost_norm_dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToDevice, *copy_stream);
	}
      }

      if (dir == 0) {
	// record the event
	hipEventRecord(ipcCopyEvent[bufferIndex][0][dim], *copy_stream);
	// send to the propcessor in the -1 direction
	comm_start(mh_send_p2p_back[bufferIndex][dim]);
      } else {
	hipEventRecord(ipcCopyEvent[bufferIndex][1][dim], *copy_stream);
	// send to the processor in the +1 direction
	comm_start(mh_send_p2p_fwd[bufferIndex][dim]);
      }
    }
  }

  void cudaColorSpinorField::commsStart(int nFace, int dir, int dagger, hipStream_t* stream_p, bool gdr) {
    recvStart(nFace, dir, dagger, stream_p, gdr);
    sendStart(nFace, dir, dagger, stream_p, gdr);
  }


  static bool complete_recv_fwd[QUDA_MAX_DIM] = { };
  static bool complete_recv_back[QUDA_MAX_DIM] = { };
  static bool complete_send_fwd[QUDA_MAX_DIM] = { };
  static bool complete_send_back[QUDA_MAX_DIM] = { };

  int cudaColorSpinorField::commsQuery(int nFace, int dir, int dagger, hipStream_t *stream_p, bool gdr) {

    int dim = dir/2;
    if (!commDimPartitioned(dim)) return 0;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but not GDR is not enabled");

    if (dir%2==0) {

      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_recv_fwd[dim]) complete_recv_fwd[dim] = comm_query(mh_recv_fwd[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_send_back[dim]) complete_send_back[dim] = comm_query(mh_send_back[bufferIndex][dim]);
      }

      if (complete_recv_fwd[dim] && complete_send_back[dim]) {
	complete_recv_fwd[dim] = false;
	complete_send_back[dim] = false;
	return 1;
      }

    } else { // dir%2 == 1

      if (comm_peer2peer_enabled(0,dim)) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_p2p_back[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	if (!complete_recv_back[dim]) complete_recv_back[dim] = comm_query(mh_recv_back[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(1,dim)) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_p2p_fwd[bufferIndex][dim]);
      } else if (gdr) {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	if (!complete_send_fwd[dim]) complete_send_fwd[dim] = comm_query(mh_send_fwd[bufferIndex][dim]);
      }

      if (complete_recv_back[dim] && complete_send_fwd[dim]) {
	complete_recv_back[dim] = false;
	complete_send_fwd[dim] = false;
	return 1;
      }

    }

    return 0;
  }

  void cudaColorSpinorField::commsWait(int nFace, int dir, int dagger, hipStream_t *stream_p, bool gdr) {
    int dim = dir / 2;
    if (!commDimPartitioned(dim)) return;
    if (gdr && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but not GDR is not enabled");

    if (dir%2==0) {

      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_recv_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][1][dim]);
      } else if (gdr) {
	comm_wait(mh_recv_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_fwd[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_send_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][0][dim]);
      } else if (gdr) {
	comm_wait(mh_send_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_back[bufferIndex][dim]);
      }
    } else {
      if (comm_peer2peer_enabled(0,dim)) {
	comm_wait(mh_recv_p2p_back[bufferIndex][dim]);
	hipEventSynchronize(ipcRemoteCopyEvent[bufferIndex][0][dim]);
      } else if (gdr) {
	comm_wait(mh_recv_rdma_back[bufferIndex][dim]);
      } else {
	comm_wait(mh_recv_back[bufferIndex][dim]);
      }

      if (comm_peer2peer_enabled(1,dim)) {
	comm_wait(mh_send_p2p_fwd[bufferIndex][dim]);
	hipEventSynchronize(ipcCopyEvent[bufferIndex][1][dim]);
      } else if (gdr) {
	comm_wait(mh_send_rdma_fwd[bufferIndex][dim]);
      } else {
	comm_wait(mh_send_fwd[bufferIndex][dim]);
      }
    }

    return;
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dim_dir, hipStream_t* stream_p)
  {
    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 recive from backwards
    if (!commDimPartitioned(dim)) return;

    if (comm_peer2peer_enabled(dir,dim)) return;
    unpackGhost(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, stream_p);
  }

  void cudaColorSpinorField::scatter(int nFace, int dagger, int dim_dir)
  {
    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 receive from backwards
    if (!commDimPartitioned(dim)) return;

    if (comm_peer2peer_enabled(dir,dim)) return;
    unpackGhost(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, &stream[dim_dir]);
  }

  void cudaColorSpinorField::scatterExtended(int nFace, int parity, int dagger, int dim_dir)
  {
    bool zero_copy = false;
    int dim = dim_dir/2;
    int dir = (dim_dir+1)%2; // dir = 1 - receive from forwards, dir == 0 receive from backwards
    if (!commDimPartitioned(dim)) return;
    unpackGhostExtended(from_face_dim_dir_h[bufferIndex][dim][dir], nFace, static_cast<QudaParity>(parity), dim, dir == 0 ? QUDA_BACKWARDS : QUDA_FORWARDS, dagger, &stream[2*dim/*+0*/], zero_copy);
  }
 
  void cudaColorSpinorField::exchangeGhost(QudaParity parity, int nFace, int dagger, const MemoryLocation *pack_destination_,
					   const MemoryLocation *halo_location_, bool gdr_send, bool gdr_recv)  const {
    if ((gdr_send || gdr_recv) && !comm_gdr_enabled()) errorQuda("Requesting GDR comms but not GDR is not enabled");
    const_cast<cudaColorSpinorField&>(*this).createComms(nFace, false);

    // first set default values to device if needed
    MemoryLocation pack_destination[2*QUDA_MAX_DIM], halo_location[2*QUDA_MAX_DIM];
    for (int i=0; i<8; i++) {
      pack_destination[i] = pack_destination_ ? pack_destination_[i] : Device;
      halo_location[i] = halo_location_ ? halo_location_[i] : Device;
    }

    // If this is set to true, then we are assuming that the send
    // buffers are in a single contiguous memory space and we çan
    // aggregate all cudaMemcpys to reduce latency.  This only applies
    // if the memory locations are all "Device".
    bool fused_pack_memcpy = true;

    // If this is set to true, then we are assuming that the send
    // buffers are in a single contiguous memory space and we çan
    // aggregate all cudaMemcpys to reduce latency.  This only applies
    // if the memory locations are all "Device".
    bool fused_halo_memcpy = true;

    // set to true if any of the ghost packing is being done to Host memory
    bool pack_host = false;

    // set to true if the final halos will be left in Host memory
    bool halo_host = false;

    void *send[2*QUDA_MAX_DIM];
    for (int d=0; d<4; d++) {
      send[2*d+0] = pack_destination[2*d+0] == Host ? my_face_dim_dir_hd[bufferIndex][d][0] : my_face_dim_dir_d[bufferIndex][d][0];
      send[2*d+1] = pack_destination[2*d+1] == Host ? my_face_dim_dir_hd[bufferIndex][d][1] : my_face_dim_dir_d[bufferIndex][d][1];
      ghost_buf[2*d+0] = halo_location[2*d+0] == Host ? from_face_dim_dir_hd[bufferIndex][d][0] : from_face_dim_dir_d[bufferIndex][d][0];
      ghost_buf[2*d+1] = halo_location[2*d+1] == Host ? from_face_dim_dir_hd[bufferIndex][d][1] : from_face_dim_dir_d[bufferIndex][d][1];
      if (pack_destination[2*d+0] != Device || pack_destination[2*d+1] != Device) fused_pack_memcpy = false;
      if (halo_location[2*d+0] != Device || halo_location[2*d+1] != Device) fused_halo_memcpy = false;

      if (pack_destination[2*d+0] == Host || pack_destination[2*d+1] == Host) pack_host = true;
      if (halo_location[2*d+0] == Host || halo_location[2*d+1] == Host) halo_host = true;
    }

    genericPackGhost(send, *this, parity, nFace, dagger, pack_destination); // FIXME - need support for asymmetric topologies

    size_t total_bytes = 0;
    for (int i=0; i<nDimComms; i++) if (comm_dim_partitioned(i)) total_bytes += 2*ghost_face_bytes[i]; // 2 for fwd/bwd

    if (!gdr_send)  {
      if (!fused_pack_memcpy) {
	for (int i=0; i<nDimComms; i++) {
	  if (comm_dim_partitioned(i)) {
	    if (pack_destination[2*i+0] == Device) qudaMemcpy(my_face_dim_dir_h[bufferIndex][i][0], my_face_dim_dir_d[bufferIndex][i][0],
							      ghost_face_bytes[i], hipMemcpyDeviceToHost);
	    if (pack_destination[2*i+1] == Device) qudaMemcpy(my_face_dim_dir_h[bufferIndex][i][1], my_face_dim_dir_d[bufferIndex][i][1],
							      ghost_face_bytes[i], hipMemcpyDeviceToHost);
	  }
	}
      } else if (total_bytes && !pack_host) {
	qudaMemcpy(my_face_h[bufferIndex], ghost_send_buffer_d[bufferIndex], total_bytes, hipMemcpyDeviceToHost);
      }
    }

    for (int i=0; i<nDimComms; i++) { // prepost receive
      if (comm_dim_partitioned(i)) {
	comm_start(gdr_recv ? mh_recv_rdma_back[bufferIndex][i] : mh_recv_back[bufferIndex][i]);
	comm_start(gdr_recv ? mh_recv_rdma_fwd[bufferIndex][i] : mh_recv_fwd[bufferIndex][i]);
      }
    }

    if (gdr_send || pack_host) hipDeviceSynchronize(); // need to make sure packing has finished before kicking off MPI

    for (int i=0; i<nDimComms; i++) {
      if (comm_dim_partitioned(i)) {
	comm_start(gdr_send ? mh_send_rdma_fwd[bufferIndex][i] : mh_send_fwd[bufferIndex][i]);
	comm_start(gdr_send ? mh_send_rdma_back[bufferIndex][i] : mh_send_back[bufferIndex][i]);
      }
    }

    for (int i=0; i<nDimComms; i++) {
      if (!comm_dim_partitioned(i)) continue;
      comm_wait(gdr_send ? mh_send_rdma_fwd[bufferIndex][i] : mh_send_fwd[bufferIndex][i]);
      comm_wait(gdr_send ? mh_send_rdma_back[bufferIndex][i] : mh_send_back[bufferIndex][i]);
      comm_wait(gdr_recv ? mh_recv_rdma_back[bufferIndex][i] : mh_recv_back[bufferIndex][i]);
      comm_wait(gdr_recv ? mh_recv_rdma_fwd[bufferIndex][i] : mh_recv_fwd[bufferIndex][i]);
    }

    if (!gdr_recv) {
      if (!fused_halo_memcpy) {
	for (int i=0; i<nDimComms; i++) {
	  if (!comm_dim_partitioned(i)) continue;
	  if (halo_location[2*i+0] == Device) qudaMemcpy(from_face_dim_dir_d[bufferIndex][i][0], from_face_dim_dir_h[bufferIndex][i][0],
							 ghost_face_bytes[i], hipMemcpyHostToDevice);
	  if (halo_location[2*i+1] == Device) qudaMemcpy(from_face_dim_dir_d[bufferIndex][i][1], from_face_dim_dir_h[bufferIndex][i][1],
							 ghost_face_bytes[i], hipMemcpyHostToDevice);
	}
      } else if (total_bytes && !halo_host) {
	qudaMemcpy(ghost_recv_buffer_d[bufferIndex], from_face_h[bufferIndex], total_bytes, hipMemcpyHostToDevice);
      }
    }

  }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

//! for composite fields:
  cudaColorSpinorField& cudaColorSpinorField::Component(const int idx) const {
    
    if (this->IsComposite()) {
      if (idx < this->CompositeDim()) {//setup eigenvector form the set
        return *(dynamic_cast<cudaColorSpinorField*>(components[idx])); 
      }
      else{
        errorQuda("Incorrect component index...");
      }
    }
    errorQuda("Cannot get requested component");
    exit(-1);
  }

//copyCuda currently cannot not work with set of spinor fields..
  void cudaColorSpinorField::CopySubset(cudaColorSpinorField &dst, const int range, const int first_element) const{
#if 0
    if (first_element < 0) errorQuda("\nError: trying to set negative first element.\n");
    if (siteSubset == QUDA_PARITY_SITE_SUBSET && this->EigvId() == -1) {
      if (first_element == 0 && range == this->EigvDim())
      {
        if (range != dst.EigvDim())errorQuda("\nError: eigenvector range to big.\n");
        checkField(dst, *this);
        copyCuda(dst, *this);
      }
      else if ((first_element+range) < this->EigvDim()) 
      {//setup eigenvector subset

        cudaColorSpinorField *eigv_subset;

        ColorSpinorParam param;

        param.nColor = nColor;
        param.nSpin = nSpin;
        param.twistFlavor = twistFlavor;
        param.precision = precision;
        param.nDim = nDim;
        param.pad = pad;
        param.siteSubset = siteSubset;
        param.siteOrder = siteOrder;
        param.fieldOrder = fieldOrder;
        param.gammaBasis = gammaBasis;
        memcpy(param.x, x, nDim*sizeof(int));
        param.create = QUDA_REFERENCE_FIELD_CREATE;
 
        param.eigv_dim  = range;
        param.eigv_id   = -1;
        param.v = (void*)((char*)v + first_element*eigv_bytes);
        param.norm = (void*)((char*)norm + first_element*eigv_norm_bytes);

        eigv_subset = new cudaColorSpinorField(param);

        //Not really needed:
        eigv_subset->eigenvectors.reserve(param.eigv_dim);
        for (int id = first_element; id < (first_element+range); id++)
        {
            param.eigv_id = id;
            eigv_subset->eigenvectors.push_back(new cudaColorSpinorField(*this, param));
        }
        checkField(dst, *eigv_subset);
        copyCuda(dst, *eigv_subset);

        delete eigv_subset;
      } else {
        errorQuda("Incorrect eigenvector dimension...");
      }
    } else{
      errorQuda("Eigenvector must be a parity spinor");
      exit(-1);
    }
#endif
  }

  void cudaColorSpinorField::getTexObjectInfo() const
  {
#ifdef USE_TEXTURE_OBJECTS
    printfQuda("\nPrint texture info for the field:\n");
    std::cout << *this;
    hipResourceDesc resDesc;
    //memset(&resDesc, 0, sizeof(resDesc));
    hipGetTextureObjectResourceDesc(&resDesc, this->Tex());
    printfQuda("\nDevice pointer: %p\n", resDesc.res.linear.devPtr);
    printfQuda("\nVolume (in bytes): %lu\n", resDesc.res.linear.sizeInBytes);
    if (resDesc.resType == hipResourceTypeLinear) printfQuda("\nResource type: linear \n");
#endif
  }

  void cudaColorSpinorField::Source(const QudaSourceType sourceType, const int st, const int s, const int c) {
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_NULL_FIELD_CREATE;

    cpuColorSpinorField tmp(param);
    tmp.Source(sourceType, st, s, c);
    *this = tmp;
  }

  void cudaColorSpinorField::PrintVector(unsigned int i) {
    ColorSpinorParam param(*this);
    param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;
    param.location = QUDA_CPU_FIELD_LOCATION;
    param.create = QUDA_NULL_FIELD_CREATE;

    cpuColorSpinorField tmp(param);
    tmp = *this;
    tmp.PrintVector(i);
  }


} // namespace quda
