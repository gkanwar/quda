#include "hip/hip_runtime.h"
#include <transfer.h>
#include <gauge_field_order.h>
#include <color_spinor_field_order.h>
#include <index_helper.cuh>
#if __COMPUTE_CAPABILITY__ >= 300
#include <generics/shfl.h>
#endif

namespace quda {

#ifdef GPU_MULTIGRID

  enum DslashType {
    DSLASH_INTERIOR,
    DSLASH_EXTERIOR,
    DSLASH_FULL
  };

  template <typename Float, int coarseSpin, int coarseColor, QudaFieldOrder csOrder, QudaGaugeFieldOrder gOrder>
  struct DslashCoarseArg {
    typedef typename colorspinor::FieldOrderCB<Float,coarseSpin,coarseColor,1,csOrder> F;
    typedef typename gauge::FieldOrder<Float,coarseColor*coarseSpin,coarseSpin,gOrder> G;

    F out;
    const F inA;
    const F inB;
    const G Y;
    const G X;
    const Float kappa;
    const int parity; // only use this for single parity fields
    const int nParity; // number of parities we're working on
    const int nFace;  // hard code to 1 for now
    const int_fastdiv X0h; // X[0]/2
    const int_fastdiv dim[5];   // full lattice dimensions
    const int commDim[4]; // whether a given dimension is partitioned or not
    const int volumeCB;

    inline DslashCoarseArg(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
			   const GaugeField &Y, const GaugeField &X, Float kappa, int parity)
      : out(const_cast<ColorSpinorField&>(out)), inA(const_cast<ColorSpinorField&>(inA)),
	inB(const_cast<ColorSpinorField&>(inB)), Y(const_cast<GaugeField&>(Y)),
	X(const_cast<GaugeField&>(X)), kappa(kappa), parity(parity),
	nParity(out.SiteSubset()), nFace(1), X0h( ((3-nParity) * out.X(0)) /2),
	dim{ (3-nParity) * out.X(0), out.X(1), out.X(2), out.X(3), out.Ndim() == 5 ? out.X(4) : 1 },
      commDim{comm_dim_partitioned(0), comm_dim_partitioned(1), comm_dim_partitioned(2), comm_dim_partitioned(3)},
      volumeCB(out.VolumeCB()/dim[4])
    {  }
  };

  /**
     @brief Helper function to determine if should halo computation
  */
  template <DslashType type>
  static __host__ __device__ bool doHalo() {
    switch(type) {
    case DSLASH_EXTERIOR:
    case DSLASH_FULL:
      return true;
    default:
      return false;
    }
  }

  /**
     @brief Helper function to determine if should interior computation
  */
  template <DslashType type>
  static __host__ __device__ bool doBulk() {
    switch(type) {
    case DSLASH_INTERIOR:
    case DSLASH_FULL:
      return true;
    default:
      return false;
    }
  }

  /**
     Compute the 4-d spatial index from the checkerboarded 1-d index at parity parity

     @param x Computed spatial index
     @param cb_index 1-d checkerboarded index
     @param X Full lattice dimensions
     @param parity Site parity
   */
  template <typename I>
  static __device__ __host__ inline void getCoordsCB(int x[], int cb_index, const I X[], const I X0h, int parity) {
    //x[3] = cb_index/(X[2]*X[1]*X[0]/2);
    //x[2] = (cb_index/(X[1]*X[0]/2)) % X[2];
    //x[1] = (cb_index/(X[0]/2)) % X[1];
    //x[0] = 2*(cb_index%(X[0]/2)) + ((x[3]+x[2]+x[1]+parity)&1);

    int za = (cb_index / X0h);
    int zb =  (za / X[1]);
    x[1] = (za - zb * X[1]);
    x[3] = (zb / X[2]);
    x[2] = (zb - x[3] * X[2]);
    int x1odd = (x[1] + x[2] + x[3] + parity) & 1;
    x[0] = (2 * cb_index + x1odd  - za * X[0]);
    return;
  }

  /**
     Applies the coarse dslash on a given parity and checkerboard site index

     @param out The result - kappa * Dslash in
     @param Y The coarse gauge field
     @param kappa Kappa value
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  extern __shared__ float s[];
  template <typename Float, int nDim, int Ns, int Nc, int Mc, int color_stride, int dim_stride, int thread_dir, int thread_dim, bool dagger, DslashType type, typename Arg>
  __device__ __host__ inline void applyDslash(complex<Float> out[], Arg &arg, int x_cb, int src_idx, int parity, int s_row, int color_block, int color_offset) {
    const int their_spinor_parity = (arg.nParity == 2) ? 1-parity : 0;

    int coord[5];
    getCoordsCB(coord, x_cb, arg.dim, arg.X0h, parity);
    coord[4] = src_idx;

#ifdef __CUDA_ARCH__
    complex<Float> *shared_sum = (complex<Float>*)s;
    if (!thread_dir) {
#endif

      //Forward gather - compute fwd offset for spinor fetch
#pragma unroll
      for(int d = thread_dim; d < nDim; d+=dim_stride) // loop over dimension
      {
	const int fwd_idx = linkIndexP1(coord, arg.dim, d);

	if ( arg.commDim[d] && (coord[d] + arg.nFace >= arg.dim[d]) ) {
	  if (doHalo<type>()) {
	    int ghost_idx = ghostFaceIndex<1>(coord, arg.dim, d, arg.nFace);

#pragma unroll
	    for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	      int c_row = color_block + color_local; // global color index
	      int row = s_row*Nc + c_row;
#pragma unroll
	      for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
#pragma unroll
		for(int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color column
		  int col = s_col*Nc + c_col + color_offset;
		  if (!dagger)
		    out[color_local] += arg.Y(d+4, parity, x_cb, row, col)
		      * arg.inA.Ghost(d, 1, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		  else
		    out[color_local] += arg.Y(d, parity, x_cb, row, col)
		      * arg.inA.Ghost(d, 1, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		}
	      }
	    }
	  }
	} else if (doBulk<type>()) {
#pragma unroll
	  for(int color_local = 0; color_local < Mc; color_local++) { //Color row
	    int c_row = color_block + color_local; // global color index
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for(int s_col = 0; s_col < Ns; s_col++) { //Spin column
#pragma unroll
	      for(int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color column
		int col = s_col*Nc + c_col + color_offset;
		if (!dagger)
		  out[color_local] += arg.Y(d+4, parity, x_cb, row, col)
		    * arg.inA(their_spinor_parity, fwd_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		else
		  out[color_local] += arg.Y(d, parity, x_cb, row, col)
		    * arg.inA(their_spinor_parity, fwd_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	    }
	  }
	}

      } // nDim

#if defined(__CUDA_ARCH__)
      if (thread_dim > 0) { // only need to write to shared memory if not master thread
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  shared_sum[((color_local * blockDim.z + threadIdx.z )*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x] = out[color_local];
	}
      }
#endif

#ifdef __CUDA_ARCH__
    } else {
#endif

      //Backward gather - compute back offset for spinor and gauge fetch
#pragma unroll
      for(int d = thread_dim; d < nDim; d+=dim_stride)
	{
	const int back_idx = linkIndexM1(coord, arg.dim, d);
	const int gauge_idx = back_idx;
	if ( arg.commDim[d] && (coord[d] - arg.nFace < 0) ) {
	  if (doHalo<type>()) {
	    const int ghost_idx = ghostFaceIndex<0>(coord, arg.dim, d, arg.nFace);
#pragma unroll
	    for (int color_local=0; color_local<Mc; color_local++) {
	      int c_row = color_block + color_local;
	      int row = s_row*Nc + c_row;
#pragma unroll
	      for (int s_col=0; s_col<Ns; s_col++)
#pragma unroll
		for (int c_col=0; c_col<Nc; c_col+=color_stride) {
		  int col = s_col*Nc + c_col + color_offset;
		  if (!dagger)
		    out[color_local] += conj(arg.Y.Ghost(d, 1-parity, ghost_idx, col, row))
		      * arg.inA.Ghost(d, 0, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		  else
		    out[color_local] += conj(arg.Y.Ghost(d+4, 1-parity, ghost_idx, col, row))
		      * arg.inA.Ghost(d, 0, their_spinor_parity, ghost_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		}
	    }
	  }
	} else if (doBulk<type>()) {
#pragma unroll
	  for(int color_local = 0; color_local < Mc; color_local++) {
	    int c_row = color_block + color_local;
	    int row = s_row*Nc + c_row;
#pragma unroll
	    for(int s_col = 0; s_col < Ns; s_col++)
#pragma unroll
	      for(int c_col = 0; c_col < Nc; c_col+=color_stride) {
		int col = s_col*Nc + c_col + color_offset;
		if (!dagger)
		  out[color_local] += conj(arg.Y(d, 1-parity, gauge_idx, col, row))
		    * arg.inA(their_spinor_parity, back_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
		else
		  out[color_local] += conj(arg.Y(d+4, 1-parity, gauge_idx, col, row))
		    * arg.inA(their_spinor_parity, back_idx + src_idx*arg.volumeCB, s_col, c_col+color_offset);
	      }
	  }
	}

      } //nDim

#if defined(__CUDA_ARCH__)

#pragma unroll
      for (int color_local=0; color_local < Mc; color_local++) {
	shared_sum[ ((color_local * blockDim.z + threadIdx.z )*blockDim.y + threadIdx.y)*blockDim.x + threadIdx.x] = out[color_local];
      }

    } // forwards / backwards thread split
#endif

#ifdef __CUDA_ARCH__ // CUDA path has to recombine the foward and backward results
    __syncthreads();

    // (colorspin * dim_stride + dim * 2 + dir)
    if (thread_dim == 0 && thread_dir == 0) {

      // full split over dimension and direction
#pragma unroll
      for (int d=1; d<dim_stride; d++) { // get remaining forward fathers (if any)
	// 4-way 1,2,3  (stride = 4)
	// 2-way 1      (stride = 2)
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  out[color_local] +=
	    shared_sum[(((color_local*blockDim.z/(2*dim_stride) + threadIdx.z/(2*dim_stride)) * 2 * dim_stride + d * 2 + 0)*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x];
	}
      }

#pragma unroll
      for (int d=0; d<dim_stride; d++) { // get all backward gathers
#pragma unroll
	for (int color_local=0; color_local < Mc; color_local++) {
	  out[color_local] +=
	    shared_sum[(((color_local*blockDim.z/(2*dim_stride) + threadIdx.z/(2*dim_stride)) * 2 * dim_stride + d * 2 + 1)*blockDim.y+threadIdx.y)*blockDim.x+threadIdx.x];
	}
      }

      // apply kappa
#pragma unroll
      for (int color_local=0; color_local<Mc; color_local++) out[color_local] *= -arg.kappa;

    }

#else // !__CUDA_ARCH__
    for (int color_local=0; color_local<Mc; color_local++) out[color_local] *= -arg.kappa;
#endif

    }

  /**
     Applies the coarse clover matrix on a given parity and
     checkerboard site index

     @param out The result out += X * in
     @param X The coarse clover field
     @param in The input field
     @param parity The site parity
     @param x_cb The checkerboarded site index
   */
  template <typename Float, int Ns, int Nc, int Mc, int color_stride, bool dagger, typename Arg>
  __device__ __host__ inline void applyClover(complex<Float> out[], Arg &arg, int x_cb, int src_idx, int parity, int s, int color_block, int color_offset) {
    const int spinor_parity = (arg.nParity == 2) ? parity : 0;

    // M is number of colors per thread
#pragma unroll
    for(int color_local = 0; color_local < Mc; color_local++) {//Color out
      int c = color_block + color_local; // global color index
      int row = s*Nc + c;
#pragma unroll
      for (int s_col = 0; s_col < Ns; s_col++) //Spin in
#pragma unroll
	for (int c_col = 0; c_col < Nc; c_col+=color_stride) { //Color in
	  //Factor of kappa and diagonal addition now incorporated in X
	  int col = s_col*Nc + c_col + color_offset;
	  if (!dagger) {
	    out[color_local] += arg.X(0, parity, x_cb, row, col)
	      * arg.inB(spinor_parity, x_cb+src_idx*arg.volumeCB, s_col, c_col+color_offset);
	  } else {
	    out[color_local] += conj(arg.X(0, parity, x_cb, col, row))
	      * arg.inB(spinor_parity, x_cb+src_idx*arg.volumeCB, s_col, c_col+color_offset);
	  }
	}
    }

  }

  //out(x) = M*in = \sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  template <typename Float, int nDim, int Ns, int Nc, int Mc, int color_stride, int dim_thread_split,
	    bool dslash, bool clover, bool dagger, DslashType type, int dir, int dim, typename Arg>
  __device__ __host__ inline void coarseDslash(Arg &arg, int x_cb, int src_idx, int parity, int s, int color_block, int color_offset)
  {
    complex <Float> out[Mc];
#pragma unroll
    for (int c=0; c<Mc; c++) out[c] = 0.0;
    if (dslash) applyDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dir,dim,dagger,type>(out, arg, x_cb, src_idx, parity, s, color_block, color_offset);
    if (doBulk<type>() && clover && dir==0 && dim==0) applyClover<Float,Ns,Nc,Mc,color_stride,dagger>(out, arg, x_cb, src_idx, parity, s, color_block, color_offset);

    if (dir==0 && dim==0) {
      const int my_spinor_parity = (arg.nParity == 2) ? parity : 0;
#pragma unroll
      for (int color_local=0; color_local<Mc; color_local++) {
#if __CUDA_ARCH__ >= 300
	// reduce down to the first group of column-split threads
	constexpr int warp_size = 32; // FIXME - this is buggy when x-dim * color_stride < 32
#pragma unroll
	for (int offset = warp_size/2; offset >= warp_size/color_stride; offset /= 2)
#if (__CUDACC_VER_MAJOR__ >= 9)
	  out[color_local] += __shfl_down_sync(WARP_CONVERGED, out[color_local], offset);
#else
	  out[color_local] += __shfl_down(out[color_local], offset);
#endif

#endif
	int c = color_block + color_local; // global color index
	if (color_offset == 0) {
	  // if not halo we just store, else we accumulate
	  if (doBulk<type>()) arg.out(my_spinor_parity, x_cb+src_idx*arg.volumeCB, s, c) = out[color_local];
	  else arg.out(my_spinor_parity, x_cb+src_idx*arg.volumeCB, s, c) += out[color_local];
	}
      }
    }
  }

  // CPU kernel for applying the coarse Dslash to a vector
  template <typename Float, int nDim, int Ns, int Nc, int Mc, bool dslash, bool clover, bool dagger, DslashType type, typename Arg>
  void coarseDslash(Arg arg)
  {
    // the fine-grain parameters mean nothing for CPU variant
    const int color_stride = 1;
    const int color_offset = 0;
    const int dim_thread_split = 1;
    const int dir = 0;
    const int dim = 0;

    for (int parity= 0; parity < arg.nParity; parity++) {
      // for full fields then set parity from loop else use arg setting
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int src_idx = 0; src_idx < arg.dim[4]; src_idx++) {
	//#pragma omp parallel for
	for(int x_cb = 0; x_cb < arg.volumeCB; x_cb++) { // 4-d volume
	  for (int s=0; s<2; s++) {
	    for (int color_block=0; color_block<Nc; color_block+=Mc) { // Mc=Nc means all colors in a thread
	      coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,dir,dim>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
	    }
	  }
	} // 4-d volumeCB
      } // src index
    } // parity

  }

  // GPU Kernel for applying the coarse Dslash to a vector
  template <typename Float, int nDim, int Ns, int Nc, int Mc, int color_stride, int dim_thread_split, bool dslash, bool clover, bool dagger, DslashType type, typename Arg>
  __global__ void coarseDslashKernel(Arg arg)
  {
    constexpr int warp_size = 32;
    const int lane_id = threadIdx.x % warp_size;
    const int warp_id = threadIdx.x / warp_size;
    const int vector_site_width = warp_size / color_stride;

    int x_cb = blockIdx.x*(blockDim.x/color_stride) + warp_id*(warp_size/color_stride) + lane_id % vector_site_width;

    const int color_offset = lane_id / vector_site_width;

    // for full fields set parity from y thread index else use arg setting
#if 0  // disable multi-src since this has a measurable impact on single src performance
    int paritySrc = blockDim.y*blockIdx.y + threadIdx.y;
    if (paritySrc >= arg.nParity * arg.dim[4]) return;
    const int src_idx = (arg.nParity == 2) ? paritySrc / 2 : paritySrc; // maybe want to swap order or source and parity for improved locality of same parity
    const int parity = (arg.nParity == 2) ? paritySrc % 2 : arg.parity;
#else
    const int src_idx = 0;
    const int parity = (arg.nParity == 2) ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
#endif

    // z thread dimension is (( s*(Nc/Mc) + color_block )*dim_thread_split + dim)*2 + dir
    int sMd = blockDim.z*blockIdx.z + threadIdx.z;
    int dir = sMd & 1;
    int sMdim = sMd >> 1;
    int dim = sMdim % dim_thread_split;
    int sM = sMdim / dim_thread_split;
    int s = sM / (Nc/Mc);
    int color_block = (sM % (Nc/Mc)) * Mc;

    if (x_cb >= arg.volumeCB) return;

    if (dir == 0) {
      if (dim == 0)      coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,0,0>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 1) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,0,1>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 2) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,0,2>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 3) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,0,3>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
    } else if (dir == 1) {
      if (dim == 0)      coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,1,0>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 1) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,1,1>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 2) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,1,2>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
      else if (dim == 3) coarseDslash<Float,nDim,Ns,Nc,Mc,color_stride,dim_thread_split,dslash,clover,dagger,type,1,3>(arg, x_cb, src_idx, parity, s, color_block, color_offset);
    }
  }

  template <typename Float, int nDim, int Ns, int Nc, int Mc, bool dslash, bool clover, bool dagger, DslashType type>
  class DslashCoarse : public Tunable {

  protected:
    ColorSpinorField &out;
    const ColorSpinorField &inA;
    const ColorSpinorField &inB;
    const GaugeField &Y;
    const GaugeField &X;
    const double kappa;
    const int parity;
    const int nParity;
    const int nSrc;

#ifdef EIGHT_WAY_WARP_SPLIT
    const int max_color_col_stride = 8;
#else
    const int max_color_col_stride = 4;
#endif
    mutable int color_col_stride;
    mutable int dim_threads;
    char *saveOut;

    long long flops() const
    {
      return ((dslash*2*nDim+clover*1)*(8*Ns*Nc*Ns*Nc)-2*Ns*Nc)*nParity*(long long)out.VolumeCB();
    }
    long long bytes() const
    {
     return (dslash||clover) * out.Bytes() + dslash*8*inA.Bytes() + clover*inB.Bytes() +
       nSrc*nParity*(dslash*Y.Bytes()*Y.VolumeCB()/(2*Y.Stride()) + clover*X.Bytes()/2);
    }
    unsigned int sharedBytesPerThread() const { return (sizeof(complex<Float>) * Mc); }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions
    bool tuneAuxDim() const { return true; } // Do tune the aux dimensions
    unsigned int minThreads() const { return color_col_stride * X.VolumeCB(); } // 4-d volume since this x threads only
    unsigned int maxBlockSize() const { return deviceProp.maxThreadsPerBlock / (dim_threads * 2 * nParity); }

    bool advanceBlockDim(TuneParam &param) const
    {
      dim3 block = param.block;
      dim3 grid = param.grid;
      bool ret = Tunable::advanceBlockDim(param);
      param.block.y = block.y; param.block.z = block.z;
      param.grid.y = grid.y; param.grid.z = grid.z;

      if (ret) { // we advanced the block.x so we're done
	return true;
      } else { // block.x (spacetime) was reset

	if (param.block.y < (unsigned int)(nParity * nSrc)) { // advance parity / 5th dimension
	  param.block.y++;
	  param.grid.y = (nParity * nSrc + param.block.y - 1) / param.block.y;
	  return true;
	} else {
	  // reset parity / 5th dimension
	  param.block.y = 1;
	  param.grid.y = nParity * nSrc;

	  // let's try to advance spin/block-color
	  while(param.block.z <= (unsigned int)(dim_threads * 2 * 2 * (Nc/Mc))) {
	    param.block.z+=dim_threads * 2;
	    if ( (dim_threads*2*2*(Nc/Mc)) % param.block.z == 0) {
	      param.grid.z = (dim_threads * 2 * 2 * (Nc/Mc)) / param.block.z;
	      break;
	    }
	  }

	  // we can advance spin/block-color since this is valid
	  if (param.block.z <= (unsigned int)(dim_threads * 2 * 2 * (Nc/Mc)) &&
	      param.block.z <= (unsigned int)deviceProp.maxThreadsDim[2] ) { //
	    return true;
	  } else { // we have run off the end so let's reset
	    param.block.z = dim_threads * 2;
	    param.grid.z = 2 * (Nc/Mc);
	    return false;
	  }
        }
      }
    }

    // FIXME: understand why this leads to slower perf and variable correctness
    //int blockStep() const { return deviceProp.warpSize/4; }
    //int blockMin() const { return deviceProp.warpSize/4; }

    // Experimental autotuning of the color column stride
    bool advanceAux(TuneParam &param) const
    {

#if __COMPUTE_CAPABILITY__ >= 300
      // we can only split the dot product on Kepler and later since we need the __shfl instruction
      if (2*param.aux.x <= max_color_col_stride && Nc % (2*param.aux.x) == 0 &&
	  param.block.x % deviceProp.warpSize == 0) {
	// An x-dimension block size that is not a multiple of the
	// warp size is incompatible with splitting the dot product
	// across the warp so we must skip this

	param.aux.x *= 2; // safe to advance
	color_col_stride = param.aux.x;

	// recompute grid size since minThreads() has now been updated
	param.grid.x = (minThreads()+param.block.x-1)/param.block.x;

	// check this grid size is valid before returning
	if (param.grid.x < (unsigned int)deviceProp.maxGridSize[0]) return true;
      }
#endif

      // reset color column stride if too large or not divisible
      param.aux.x = 1;
      color_col_stride = param.aux.x;

      // recompute grid size since minThreads() has now been updated
      param.grid.x = (minThreads()+param.block.x-1)/param.block.x;

      if (2*param.aux.y <= nDim) {
	param.aux.y *= 2;
	dim_threads = param.aux.y;

	// need to reset z-block/grid size/shared_bytes since dim_threads has changed
	param.block.z = dim_threads * 2;
	param.grid.z = 2* (Nc / Mc);

	param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);

	return true;
      } else {
	param.aux.y = 1;
	dim_threads = param.aux.y;

	// need to reset z-block/grid size/shared_bytes since
	// dim_threads has changed.  Strictly speaking this isn't needed
	// since this is the outer dimension to tune, but would be
	// needed if we added an aux.z tuning dimension
	param.block.z = dim_threads * 2;
	param.grid.z = 2* (Nc / Mc);

	param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	  sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);

	return false;
      }
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      param.aux = make_int4(1,1,1,1);
      color_col_stride = param.aux.x;
      dim_threads = param.aux.y;

      Tunable::initTuneParam(param);
      param.block.y = 1;
      param.grid.y = nParity * nSrc;
      param.block.z = dim_threads * 2;
      param.grid.z = 2*(Nc/Mc);
      param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      param.aux = make_int4(1,1,1,1);
      color_col_stride = param.aux.x;
      dim_threads = param.aux.y;

      Tunable::defaultTuneParam(param);
      // ensure that the default x block size is divisible by the warpSize
      param.block.x = deviceProp.warpSize;
      param.grid.x = (minThreads()+param.block.x-1)/param.block.x;
      param.block.y = 1;
      param.grid.y = nParity * nSrc;
      param.block.z = dim_threads * 2;
      param.grid.z = 2*(Nc/Mc);
      param.shared_bytes = sharedBytesPerThread()*param.block.x*param.block.y*param.block.z > sharedBytesPerBlock(param) ?
	sharedBytesPerThread()*param.block.x*param.block.y*param.block.z : sharedBytesPerBlock(param);
    }

  public:
    inline DslashCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
			const GaugeField &Y, const GaugeField &X, double kappa, int parity, MemoryLocation *halo_location)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity),
      nParity(out.SiteSubset()), nSrc(out.Ndim()==5 ? out.X(4) : 1)
    {
      strcpy(aux, out.AuxString());
      strcat(aux, comm_dim_partitioned_string());

      // record the location of where each pack buffer is in [2*dim+dir] ordering
      // 0 - no packing
      // 1 - pack to local GPU memory
      // 2 - pack to local mapped CPU memory
      // 3 - pack to remote mapped GPU memory
      switch(type) {
      case DSLASH_INTERIOR: strcat(aux,",interior"); break;
      case DSLASH_EXTERIOR: strcat(aux,",exterior"); break;
      case DSLASH_FULL:     strcat(aux,",full"); break;
      }

      if (doHalo<type>()) {
	char label[15] = ",halo=";
	for (int dim=0; dim<4; dim++) {
	  for (int dir=0; dir<2; dir++) {
	    label[2*dim+dir+6] = !comm_dim_partitioned(dim) ? '0' : halo_location[2*dim+dir] == Device ? '1' : halo_location[2*dim+dir] == Host ? '2' : '3';
	  }
	}
	label[14] = '\0';
	strcat(aux,label);
      }
    }
    virtual ~DslashCoarse() { }

    inline void apply(const hipStream_t &stream) {

      if (out.Location() == QUDA_CPU_FIELD_LOCATION) {

	if (out.FieldOrder() != QUDA_SPACE_SPIN_COLOR_FIELD_ORDER || Y.FieldOrder() != QUDA_QDP_GAUGE_ORDER)
	  errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", inA.FieldOrder(), Y.FieldOrder());

	DslashCoarseArg<Float,Ns,Nc,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER,QUDA_QDP_GAUGE_ORDER> arg(out, inA, inB, Y, X, (Float)kappa, parity);
	coarseDslash<Float,nDim,Ns,Nc,Mc,dslash,clover,dagger,type>(arg);
      } else {

        const TuneParam &tp = tuneLaunch(*this, getTuning(), QUDA_VERBOSE /*getVerbosity()*/);

	if (out.FieldOrder() != QUDA_FLOAT2_FIELD_ORDER || Y.FieldOrder() != QUDA_FLOAT2_GAUGE_ORDER)
	  errorQuda("Unsupported field order colorspinor=%d gauge=%d combination\n", inA.FieldOrder(), Y.FieldOrder());

	DslashCoarseArg<Float,Ns,Nc,QUDA_FLOAT2_FIELD_ORDER,QUDA_FLOAT2_GAUGE_ORDER> arg(out, inA, inB, Y, X, (Float)kappa, parity);

	switch (tp.aux.y) { // dimension gather parallelisation
	case 1:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,1,1,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,2,1,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,4,1,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#ifdef EIGHT_WAY_WARP_SPLIT
	  case 8:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,8,1,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#endif
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	case 2:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,1,2,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,2,2,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,4,2,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#ifdef EIGHT_WAY_WARP_SPLIT
	  case 8:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,8,2,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#endif
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	case 4:
	  switch (tp.aux.x) { // this is color_col_stride
	  case 1:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,1,4,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 2:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,2,4,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
	  case 4:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,4,4,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#ifdef EIGHT_WAY_WARP_SPLIT
	  case 8:
	    coarseDslashKernel<Float,nDim,Ns,Nc,Mc,8,4,dslash,clover,dagger,type> <<<tp.grid,tp.block,tp.shared_bytes,stream>>>(arg);
	    break;
#endif
	  default:
	    errorQuda("Color column stride %d not valid", tp.aux.x);
	  }
	  break;
	default:
	  errorQuda("Invalid dimension thread splitting %d", tp.aux.y);
	}
      }
    }

    TuneKey tuneKey() const {
      return TuneKey(out.VolString(), typeid(*this).name(), aux);
    }

    void preTune() {
      saveOut = new char[out.Bytes()];
      hipMemcpy(saveOut, out.V(), out.Bytes(), hipMemcpyDeviceToHost);
    }

    void postTune()
    {
      hipMemcpy(out.V(), saveOut, out.Bytes(), hipMemcpyHostToDevice);
      delete[] saveOut;
    }

  };


  template <typename Float, int coarseColor, int coarseSpin>
  inline void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
			  const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash,
			  bool clover, bool dagger, DslashType type, MemoryLocation *halo_location) {

    const int colors_per_thread = 1;
    const int nDim = 4;

    if (dagger) {
      if (dslash) {
	if (clover) {
	  if (type == DSLASH_FULL) {
	    DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,true,true,true,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	    dslash.apply(0);
	  } else { errorQuda("Dslash type %d not instantiated", type); }
	} else {
	  if (type == DSLASH_FULL) {
	    DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,true,false,true,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	    dslash.apply(0);
	  } else { errorQuda("Dslash type %d not instantiated", type); }
	}
      } else {
	if (type == DSLASH_EXTERIOR) errorQuda("Cannot call halo on pure clover kernel");
	if (clover) {
	  DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,false,true,true,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	  dslash.apply(0);
	} else {
	  errorQuda("Unsupported dslash=false clover=false");
	}
      }
    } else {
      if (dslash) {
	if (clover) {
	  if (type == DSLASH_FULL) {
	    DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,true,true,false,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	    dslash.apply(0);
	  } else { errorQuda("Dslash type %d not instantiated", type); }
	} else {
	  if (type == DSLASH_FULL) {
	    DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,true,false,false,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	    dslash.apply(0);
	  } else { errorQuda("Dslash type %d not instantiated", type); }
	}
      } else {
	if (type == DSLASH_EXTERIOR) errorQuda("Cannot call halo on pure clover kernel");
	if (clover) {
	  DslashCoarse<Float,nDim,coarseSpin,coarseColor,colors_per_thread,false,true,false,DSLASH_FULL> dslash(out, inA, inB, Y, X, kappa, parity, halo_location);
	  dslash.apply(0);
	} else {
	  errorQuda("Unsupported dslash=false clover=false");
	}
      }
    }
  }

  // template on the number of coarse colors
  template <typename Float>
  inline void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
			  const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash,
			  bool clover, bool dagger, DslashType type, MemoryLocation *halo_location) {

    if (Y.FieldOrder() != X.FieldOrder())
      errorQuda("Field order mismatch Y = %d, X = %d", Y.FieldOrder(), X.FieldOrder());

    if (inA.FieldOrder() != out.FieldOrder())
      errorQuda("Field order mismatch inA = %d, out = %d", inA.FieldOrder(), out.FieldOrder());

    if (inA.Nspin() != 2)
      errorQuda("Unsupported number of coarse spins %d\n",inA.Nspin());

    if (inA.Ncolor() == 2) {
      ApplyCoarse<Float,2,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
#if 0
    } else if (inA.Ncolor() == 4) {
      ApplyCoarse<Float,4,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
    } else if (inA.Ncolor() == 8) {
      ApplyCoarse<Float,8,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
    } else if (inA.Ncolor() == 12) {
      ApplyCoarse<Float,12,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
    } else if (inA.Ncolor() == 16) {
      ApplyCoarse<Float,16,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
    } else if (inA.Ncolor() == 20) {
      ApplyCoarse<Float,20,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
#endif
    } else if (inA.Ncolor() == 24) {
      ApplyCoarse<Float,24,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
#if 0
    } else if (inA.Ncolor() == 28) {
      ApplyCoarse<Float,28,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
#endif
    } else if (inA.Ncolor() == 32) {
      ApplyCoarse<Float,32,2>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, type, halo_location);
    } else {
      errorQuda("Unsupported number of coarse dof %d\n", Y.Ncolor());
    }
  }

  // this is the Worker pointer that may have issue additional work
  // while we're waiting on communication to finish
  namespace dslash {
    extern Worker* aux_worker;
  }

#endif // GPU_MULTIGRID

  enum DslashCoarsePolicy {
    DSLASH_COARSE_BASIC,          // stage both sends and recvs in host memory using memcpys
    DSLASH_COARSE_ZERO_COPY_PACK, // zero copy write pack buffers
    DSLASH_COARSE_ZERO_COPY_READ, // zero copy read halos in dslash kernel
    DSLASH_COARSE_ZERO_COPY,      // full zero copy
    DSLASH_COARSE_GDR_SEND,       // GDR send
    DSLASH_COARSE_GDR_RECV,       // GDR recv
    DSLASH_COARSE_GDR,             // full GDR
    DSLASH_COARSE_ZERO_COPY_PACK_GDR_RECV, // zero copy write and GDR recv
    DSLASH_COARSE_GDR_SEND_ZERO_COPY_READ // GDR send and zero copy read
  };

  struct DslashCoarseLaunch {

    ColorSpinorField &out;
    const ColorSpinorField &inA;
    const ColorSpinorField &inB;
    const GaugeField &Y;
    const GaugeField &X;
    double kappa;
    int parity;
    bool dslash;
    bool clover;
    bool dagger;

    inline DslashCoarseLaunch(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
			      const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover, bool dagger)
      : out(out), inA(inA), inB(inB), Y(Y), X(X), kappa(kappa), parity(parity), dslash(dslash), clover(clover), dagger(dagger) { }

    /**
       @brief Execute the coarse dslash using the given policy
     */
    inline void operator()(DslashCoarsePolicy policy) {
#ifdef GPU_MULTIGRID
      if (inA.V() == out.V()) errorQuda("Aliasing pointers");

      // check all precisions match
      QudaPrecision precision = checkPrecision(out, inA, inB, Y, X);

      // check all locations match
      checkLocation(out, inA, inB, Y, X);

      MemoryLocation pack_destination[2*QUDA_MAX_DIM]; // where we will pack the ghost buffer to
      MemoryLocation halo_location[2*QUDA_MAX_DIM]; // where we load the halo from
      for (int i=0; i<2*QUDA_MAX_DIM; i++) {
	pack_destination[i] = (policy == DSLASH_COARSE_ZERO_COPY_PACK || policy == DSLASH_COARSE_ZERO_COPY ||
			       policy == DSLASH_COARSE_ZERO_COPY_PACK_GDR_RECV) ? Host : Device;
	halo_location[i] = (policy == DSLASH_COARSE_ZERO_COPY_READ || policy == DSLASH_COARSE_ZERO_COPY ||
			    policy == DSLASH_COARSE_GDR_SEND_ZERO_COPY_READ) ? Host : Device;
      }
      bool gdr_send = (policy == DSLASH_COARSE_GDR_SEND || policy == DSLASH_COARSE_GDR ||
		       policy == DSLASH_COARSE_GDR_SEND_ZERO_COPY_READ) ? true : false;
      bool gdr_recv = (policy == DSLASH_COARSE_GDR_RECV || policy == DSLASH_COARSE_GDR ||
		       policy == DSLASH_COARSE_ZERO_COPY_PACK_GDR_RECV) ? true : false;

      if (dslash && comm_partitioned()) {
	const int nFace = 1;
	inA.exchangeGhost((QudaParity)(1-parity), nFace, dagger, pack_destination, halo_location, gdr_send, gdr_recv);
      }

      if (dslash::aux_worker) dslash::aux_worker->apply(0);

      if (precision == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
	ApplyCoarse<double>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, DSLASH_FULL, halo_location);
	//if (dslash && comm_partitioned()) ApplyCoarse<double>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, true, halo_location);
#else
	errorQuda("Double precision multigrid has not been enabled");
#endif
      } else if (precision == QUDA_SINGLE_PRECISION) {
	ApplyCoarse<float>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, DSLASH_FULL, halo_location);
	//if (dslash && comm_partitioned()) ApplyCoarse<float>(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger, true, halo_location);
      } else {
	errorQuda("Unsupported precision %d\n", Y.Precision());
      }

      if (dslash && comm_partitioned()) inA.bufferIndex = (1 - inA.bufferIndex);
#else
      errorQuda("Multigrid has not been built");
#endif
    }

  };

  // hooks into tune.cpp variables for policy tuning
  typedef std::map<TuneKey, TuneParam> map;
  const map& getTuneCache();

  void disableProfileCount();
  void enableProfileCount();
  void setPolicyTuning(bool);

  static bool dslash_init = false;
  static std::vector<DslashCoarsePolicy> policy;
  static int config = 0; // 2-bit number used to record the machine config (p2p / gdr) and if this changes we will force a retune

 class DslashCoarsePolicyTune : public Tunable {

   DslashCoarseLaunch &dslash;

   bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
   bool tuneAuxDim() const { return true; } // Do tune the aux dimensions.
   unsigned int sharedBytesPerThread() const { return 0; }
   unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0; }

 public:
   inline DslashCoarsePolicyTune(DslashCoarseLaunch &dslash) : dslash(dslash)
   {
      strcpy(aux,"policy,");
      if (dslash.dslash) strcat(aux,"dslash");
      strcat(aux, dslash.clover ? "clover," : ",");
      strcat(aux,dslash.inA.AuxString());
      strcat(aux,comm_dim_partitioned_string());

      if (!dslash_init) {
	policy.reserve(9);
	static char *dslash_policy_env = getenv("QUDA_ENABLE_DSLASH_COARSE_POLICY");

	if (dslash_policy_env) { // set the policies to tune for explicitly
	  std::stringstream policy_list(dslash_policy_env);

	  int policy_;
	  while (policy_list >> policy_) {
	    DslashCoarsePolicy dslash_policy = static_cast<DslashCoarsePolicy>(policy_);

	    // check this is a valid policy choice
	    if ( (dslash_policy == DSLASH_COARSE_GDR_SEND || dslash_policy == DSLASH_COARSE_GDR_RECV ||
		  dslash_policy == DSLASH_COARSE_GDR || dslash_policy == DSLASH_COARSE_ZERO_COPY_PACK_GDR_RECV ||
		  dslash_policy == DSLASH_COARSE_GDR_SEND_ZERO_COPY_READ) && !comm_gdr_enabled() ) {
	      errorQuda("Cannot select a GDR policy %d unless QUDA_ENABLE_GDR is set", dslash_policy);
	    }

	    policy.push_back(static_cast<DslashCoarsePolicy>(policy_));
	    if (policy_list.peek() == ',') policy_list.ignore();
	  }
	} else {
	  policy.push_back(DSLASH_COARSE_BASIC);
	  policy.push_back(DSLASH_COARSE_ZERO_COPY_PACK);
	  policy.push_back(DSLASH_COARSE_ZERO_COPY_READ);
	  policy.push_back(DSLASH_COARSE_ZERO_COPY);
	  if (comm_gdr_enabled()) {
	    policy.push_back(DSLASH_COARSE_GDR_SEND);
	    policy.push_back(DSLASH_COARSE_GDR_RECV);
	    policy.push_back(DSLASH_COARSE_GDR);
	    policy.push_back(DSLASH_COARSE_ZERO_COPY_PACK_GDR_RECV);
	    policy.push_back(DSLASH_COARSE_GDR_SEND_ZERO_COPY_READ);
	  }
	}

	config += comm_peer2peer_enabled_global();
	config += comm_gdr_enabled() * 2;
	dslash_init = true;
      }

      // before we do policy tuning we must ensure the kernel
      // constituents have been tuned since we can't do nested tuning
      if (getTuning() && getTuneCache().find(tuneKey()) == getTuneCache().end()) {
	disableProfileCount();
	for (auto &i : policy) dslash(i);
	enableProfileCount();
	setPolicyTuning(true);
      }
    }

   virtual ~DslashCoarsePolicyTune() { setPolicyTuning(false); }

   inline void apply(const hipStream_t &stream) {
     TuneParam tp = tuneLaunch(*this, getTuning(), QUDA_DEBUG_VERBOSE /*getVerbosity()*/);

     if (config != tp.aux.y) {
       errorQuda("Machine configuration (P2P/GDR=%d) changed since tunecache was created (P2P/GDR=%d).  Please delete "
		 "this file or set the QUDA_RESOURCE_PATH environment variable to point to a new path.",
		 config, tp.aux.y);
     }

     if (tp.aux.x >= (int)policy.size()) errorQuda("Requested policy that is outside of range");
     dslash(policy[tp.aux.x]);
   }

   int tuningIter() const { return 10; }

   bool advanceAux(TuneParam &param) const
   {
     if ((unsigned)param.aux.x < policy.size()-1) {
       param.aux.x++;
       return true;
     } else {
       param.aux.x = 0;
       return false;
     }
   }

   bool advanceTuneParam(TuneParam &param) const { return advanceAux(param); }

   void initTuneParam(TuneParam &param) const  {
     Tunable::initTuneParam(param);
     param.aux.x = 0; param.aux.y = config; param.aux.z = 0; param.aux.w = 0;
   }

   void defaultTuneParam(TuneParam &param) const  {
     Tunable::defaultTuneParam(param);
     param.aux.x = 0; param.aux.y = config; param.aux.z = 0; param.aux.w = 0;
   }

   TuneKey tuneKey() const {
     return TuneKey(dslash.inA.VolString(), typeid(*this).name(), aux);
   }

   long long flops() const {
     int nDim = 4;
     int Ns = dslash.inA.Nspin();
     int Nc = dslash.inA.Ncolor();
     int nParity = dslash.inA.SiteSubset();
     int volumeCB = dslash.inA.VolumeCB();
     return ((dslash.dslash*2*nDim+dslash.clover*1)*(8*Ns*Nc*Ns*Nc)-2*Ns*Nc)*nParity*volumeCB;
   }

   long long bytes() const {
     int nParity = dslash.inA.SiteSubset();
     return (dslash.dslash||dslash.clover) * dslash.out.Bytes() +
       dslash.dslash*8*dslash.inA.Bytes() + dslash.clover*dslash.inB.Bytes() +
       nParity*(dslash.dslash*dslash.Y.Bytes()*dslash.Y.VolumeCB()/(2*dslash.Y.Stride())
		+ dslash.clover*dslash.X.Bytes()/2);
     // multiply Y by volume / stride to correct for pad
   }
  };


  //Apply the coarse Dirac matrix to a coarse grid vector
  //out(x) = M*in = X*in - kappa*\sum_mu Y_{-\mu}(x)in(x+mu) + Y^\dagger_mu(x-mu)in(x-mu)
  //  or
  //out(x) = M^dagger*in = X^dagger*in - kappa*\sum_mu Y^\dagger_{-\mu}(x)in(x+mu) + Y_mu(x-mu)in(x-mu)
  //Uses the kappa normalization for the Wilson operator.
  void ApplyCoarse(ColorSpinorField &out, const ColorSpinorField &inA, const ColorSpinorField &inB,
	           const GaugeField &Y, const GaugeField &X, double kappa, int parity, bool dslash, bool clover, bool dagger) {

    DslashCoarseLaunch Dslash(out, inA, inB, Y, X, kappa, parity, dslash, clover, dagger);

    DslashCoarsePolicyTune policy(Dslash);
    policy.apply(0);

  }//ApplyCoarse


} // namespace quda
