#include "hip/hip_runtime.h"
#include <color_spinor_field.h>
#include <color_spinor_field_order.h>
#include <tune_quda.h>
#include <typeinfo>

#include <multigrid_helper.cuh>

namespace quda {

#ifdef GPU_MULTIGRID
  using namespace quda::colorspinor;
  
  /** 
      Kernel argument struct
  */
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, QudaFieldOrder order>
  struct ProlongateArg {
    FieldOrderCB<Float,fineSpin,fineColor,1,order> out;
    const FieldOrderCB<Float,coarseSpin,coarseColor,1,order> in;
    const FieldOrderCB<Float,fineSpin,fineColor,coarseColor,order> V;
    const int *geo_map;  // need to make a device copy of this
    const spin_mapper<fineSpin,coarseSpin> spin_map;
    const int parity; // the parity of the output field (if single parity)
    const int nParity; // number of parities of input fine field

    ProlongateArg(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &V,
		  const int *geo_map,  const int parity)
      : out(out), in(in), V(V), geo_map(geo_map), spin_map(), parity(parity), nParity(out.SiteSubset()) { }

    ProlongateArg(const ProlongateArg<Float,fineSpin,fineColor,coarseSpin,coarseColor,order> &arg)
      : out(arg.out), in(arg.in), V(arg.V), geo_map(arg.geo_map), spin_map(),
	parity(arg.parity), nParity(arg.nParity) { }
  };

  /**
     Applies the grid prolongation operator (coarse to fine)
  */
  template <typename Float, int fineSpin, int coarseColor, class Coarse, typename S>
  __device__ __host__ inline void prolongate(complex<Float> out[fineSpin*coarseColor], const Coarse &in, 
					     int parity, int x_cb, const int *geo_map, const S& spin_map, int fineVolumeCB) {
    int x = parity*fineVolumeCB + x_cb;
    int x_coarse = geo_map[x];
    int parity_coarse = (x_coarse >= in.VolumeCB()) ? 1 : 0;
    int x_coarse_cb = x_coarse - parity_coarse*in.VolumeCB();

#pragma unroll
    for (int s=0; s<fineSpin; s++) {
#pragma unroll
      for (int c=0; c<coarseColor; c++) {
	out[s*coarseColor+c] = in(parity_coarse, x_coarse_cb, spin_map(s), c);
      }
    }
  }

  /**
     Rotates from the coarse-color basis into the fine-color basis.  This
     is the second step of applying the prolongator.
  */
  template <typename Float, int fineSpin, int fineColor, int coarseColor, int fine_colors_per_thread,
	    class FineColor, class Rotator>
  __device__ __host__ inline void rotateFineColor(FineColor &out, const complex<Float> in[fineSpin*coarseColor],
						  const Rotator &V, int parity, int nParity, int x_cb, int fine_color_block) {
    const int spinor_parity = (nParity == 2) ? parity : 0;
    const int v_parity = (V.Nparity() == 2) ? parity : 0;

    constexpr int color_unroll = 2;

#pragma unroll
    for (int s=0; s<fineSpin; s++)
#pragma unroll
      for (int fine_color_local=0; fine_color_local<fine_colors_per_thread; fine_color_local++)
	out(spinor_parity, x_cb, s, fine_color_block+fine_color_local) = 0.0; // global fine color index
    
#pragma unroll
    for (int s=0; s<fineSpin; s++) {
#pragma unroll
      for (int fine_color_local=0; fine_color_local<fine_colors_per_thread; fine_color_local++) {
	int i = fine_color_block + fine_color_local; // global fine color index

	complex<Float> partial[color_unroll];
#pragma unroll
	for (int k=0; k<color_unroll; k++) partial[k] = 0.0;

#pragma unroll
	for (int j=0; j<coarseColor; j+=color_unroll) {
	  // V is a ColorMatrixField with internal dimensions Ns * Nc * Nvec
#pragma unroll
	  for (int k=0; k<color_unroll; k++)
	    partial[k] += V(v_parity, x_cb, s, i, j+k) * in[s*coarseColor + j + k];
	}

#pragma unroll
	for (int k=0; k<color_unroll; k++) out(spinor_parity, x_cb, s, i) += partial[k];
      }
    }

  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread, typename Arg>
  void Prolongate(Arg &arg) {
    for (int parity=0; parity<arg.nParity; parity++) {
      parity = (arg.nParity == 2) ? parity : arg.parity;

      for (int x_cb=0; x_cb<arg.out.VolumeCB(); x_cb++) {
	complex<Float> tmp[fineSpin*coarseColor];
	prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.VolumeCB());
	for (int fine_color_block=0; fine_color_block<fineColor; fine_color_block+=fine_colors_per_thread) {
	  rotateFineColor<Float,fineSpin,fineColor,coarseColor,fine_colors_per_thread>
	    (arg.out, tmp, arg.V, parity, arg.nParity, x_cb, fine_color_block);
	}
      }
    }
  }

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread, typename Arg>
  __global__ void ProlongateKernel(Arg arg) {
    int x_cb = blockIdx.x*blockDim.x + threadIdx.x;
    int parity = arg.nParity == 2 ? blockDim.y*blockIdx.y + threadIdx.y : arg.parity;
    if (x_cb >= arg.out.VolumeCB()) return;

    int fine_color_block = (blockDim.z*blockIdx.z + threadIdx.z) * fine_colors_per_thread;
    if (fine_color_block >= fineColor) return;

    complex<Float> tmp[fineSpin*coarseColor];
    prolongate<Float,fineSpin,coarseColor>(tmp, arg.in, parity, x_cb, arg.geo_map, arg.spin_map, arg.out.VolumeCB());
    rotateFineColor<Float,fineSpin,fineColor,coarseColor,fine_colors_per_thread>
      (arg.out, tmp, arg.V, parity, arg.nParity, x_cb, fine_color_block);
  }
  
  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor, int fine_colors_per_thread>
  class ProlongateLaunch : public TunableVectorYZ {

  protected:
    ColorSpinorField &out;
    const ColorSpinorField &in;
    const ColorSpinorField &V;
    const int *fine_to_coarse;
    int parity;
    QudaFieldLocation location;
    char vol[TuneKey::volume_n];

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return out.VolumeCB(); } // fine parity is the block y dimension

  public:
    ProlongateLaunch(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &V,
		     const int *fine_to_coarse, int parity)
      : TunableVectorYZ(out.SiteSubset(), fineColor/fine_colors_per_thread), out(out), in(in), V(V),
	fine_to_coarse(fine_to_coarse), parity(parity), location(checkLocation(out, in, V))
    {
      strcpy(vol, out.VolString());
      strcat(vol, ",");
      strcat(vol, in.VolString());

      strcpy(aux, out.AuxString());
      strcat(aux, ",");
      strcat(aux, in.AuxString());
    }

    virtual ~ProlongateLaunch() { }

    void apply(const hipStream_t &stream) {
      if (location == QUDA_CPU_FIELD_LOCATION) {
	if (out.FieldOrder() == QUDA_SPACE_SPIN_COLOR_FIELD_ORDER) {
	  ProlongateArg<Float,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_SPACE_SPIN_COLOR_FIELD_ORDER>
	    arg(out, in, V, fine_to_coarse, parity);
	  Prolongate<Float,fineSpin,fineColor,coarseSpin,coarseColor,fine_colors_per_thread>(arg);
	} else {
	  errorQuda("Unsupported field order %d", out.FieldOrder());
	}
      } else {
	if (out.FieldOrder() == QUDA_FLOAT2_FIELD_ORDER) {
	  TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
	  ProlongateArg<Float,fineSpin,fineColor,coarseSpin,coarseColor,QUDA_FLOAT2_FIELD_ORDER>
	    arg(out, in, V, fine_to_coarse, parity);
	  ProlongateKernel<Float,fineSpin,fineColor,coarseSpin,coarseColor,fine_colors_per_thread>
	    <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
	} else {
	  errorQuda("Unsupported field order %d", out.FieldOrder());
	}
      }
    }

    TuneKey tuneKey() const { return TuneKey(vol, typeid(*this).name(), aux); }

    long long flops() const { return 8 * fineSpin * fineColor * coarseColor * out.SiteSubset()*out.VolumeCB(); }

    long long bytes() const {
      size_t v_bytes = V.Bytes() / (V.SiteSubset() == out.SiteSubset() ? 1 : 2);
      return in.Bytes() + out.Bytes() + v_bytes + out.SiteSubset()*out.VolumeCB()*sizeof(int);
    }

  };

  template <typename Float, int fineSpin, int fineColor, int coarseSpin, int coarseColor>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  const int *fine_to_coarse, int parity) {

    // for all grids use 1 color per thread
    constexpr int fine_colors_per_thread = 1;

    ProlongateLaunch<Float, fineSpin, fineColor, coarseSpin, coarseColor, fine_colors_per_thread>
      prolongator(out, in, v, fine_to_coarse, parity);
    prolongator.apply(0);

    if (checkLocation(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
  }


  template <typename Float, int fineSpin>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int nVec, const int *fine_to_coarse, const int *spin_map, int parity) {

    if (in.Nspin() != 2) errorQuda("Coarse spin %d is not supported", in.Nspin());
    const int coarseSpin = 2;

    // first check that the spin_map matches the spin_mapper
    spin_mapper<fineSpin,coarseSpin> mapper;
    for (int s=0; s<fineSpin; s++) 
      if (mapper(s) != spin_map[s]) errorQuda("Spin map does not match spin_mapper");

    if (out.Ncolor() == 3) {
      const int fineColor = 3;
      if (nVec == 2) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,2>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 4) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,4>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 24) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 32) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (out.Ncolor() == 2) {
      const int fineColor = 2;
      if (nVec == 2) { // these are probably only for debugging only
	Prolongate<Float,fineSpin,fineColor,coarseSpin,2>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 4) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,4>(out, in, v, fine_to_coarse, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (out.Ncolor() == 24) {
      const int fineColor = 24;
      if (nVec == 24) { // to keep compilation under control coarse grids have same or more colors
	Prolongate<Float,fineSpin,fineColor,coarseSpin,24>(out, in, v, fine_to_coarse, parity);
      } else if (nVec == 32) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else if (out.Ncolor() == 32) {
      const int fineColor = 32;
      if (nVec == 32) {
	Prolongate<Float,fineSpin,fineColor,coarseSpin,32>(out, in, v, fine_to_coarse, parity);
      } else {
	errorQuda("Unsupported nVec %d", nVec);
      }
    } else {
      errorQuda("Unsupported nColor %d", out.Ncolor());
    }
  }

  template <typename Float>
  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map, int parity) {

    if (out.Nspin() == 4) {
      Prolongate<Float,4>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
    } else if (out.Nspin() == 2) {
      Prolongate<Float,2>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#ifdef GPU_STAGGERED_DIRAC
    } else if (out.Nspin() == 1) {
      Prolongate<Float,1>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#endif
    } else {
      errorQuda("Unsupported nSpin %d", out.Nspin());
    }
  }

#endif // GPU_MULTIGRID

  void Prolongate(ColorSpinorField &out, const ColorSpinorField &in, const ColorSpinorField &v,
		  int Nvec, const int *fine_to_coarse, const int *spin_map, int parity) {
#ifdef GPU_MULTIGRID
    if (out.FieldOrder() != in.FieldOrder() || out.FieldOrder() != v.FieldOrder())
      errorQuda("Field orders do not match (out=%d, in=%d, v=%d)", 
		out.FieldOrder(), in.FieldOrder(), v.FieldOrder());

    QudaPrecision precision = checkPrecision(out, in, v);

    if (precision == QUDA_DOUBLE_PRECISION) {
#ifdef GPU_MULTIGRID_DOUBLE
      Prolongate<double>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
#else
      errorQuda("Double precision multigrid has not been enabled");
#endif
    } else if (precision == QUDA_SINGLE_PRECISION) {
      Prolongate<float>(out, in, v, Nvec, fine_to_coarse, spin_map, parity);
    } else {
      errorQuda("Unsupported precision %d", out.Precision());
    }

    if (checkLocation(out, in, v) == QUDA_CUDA_FIELD_LOCATION) checkCudaError();
#else
    errorQuda("Multigrid has not been built");
#endif
  }

} // end namespace quda
